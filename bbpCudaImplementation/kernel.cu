#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <Windows.h>
#include <deque>
#include <atomic>
#include <fstream>
#include <iostream>
#include <filesystem>
#include <string>

#define TYPE unsigned long long
#define INT_64 unsigned long long

const int totalGpus = 2;

//warpsize is 32 so optimal value is probably always a multiple of 32
const int threadCountPerBlock = 128;
//this is more difficult to optimize but seems to not like odd numbers
const int blockCount = 560;

__device__ __constant__ const TYPE baseSystem = 1024;
__device__  __constant__ const int baseExpOf2 = 10;

__device__ const int typeSize = sizeof(TYPE) * 8 - 1;
__device__ const TYPE multiplyModCond = 0x4000000000000000;//2^62
__device__  __constant__ const INT_64 int64MaxBit = 0x8000000000000000;

__device__ int printOnce = 0;

struct sJ {
	double s1 = 0.0, s4 = 0.0, s5 = 0.0, s6 = 0.0;
	double s4k1 = 0.0, s4k3 = 0.0, s10k1 = 0.0, s10k3 = 0.0, s10k5 = 0.0, s10k7 = 0.0, s10k9 = 0.0;
};

typedef struct {
	volatile INT_64 *currentProgress;
	volatile INT_64 *deviceProg;
	sJ previousCache;
	double previousTime;
	sJ status[totalGpus];
	volatile INT_64 nextStrideBegin[totalGpus];
	TYPE maxProgress;
	volatile int quit = 0;
	hipError_t error;
	clock_t begin;
	volatile std::atomic<int> dataWritten;
} PROGRESSDATA, *PPROGRESSDATA;

typedef struct {
	sJ output;
	INT_64 digit;
	INT_64 beginFrom;
	int gpu = 0;
	int totalGpus = 0;
	int size = 0;
	hipError_t error;
	volatile INT_64 *deviceProg;
	sJ * status;
	volatile INT_64 * nextStrideBegin;
	volatile std::atomic<int> * dataWritten;
} BBPLAUNCHERDATA, *PBBPLAUNCHERDATA;

PPROGRESSDATA setupProgress();
DWORD WINAPI progressCheck(LPVOID data);
DWORD WINAPI cudaBbpLauncher(LPVOID dataV);

//adds all elements of addend and augend, storing in addend
__device__ __host__ void sJAdd(sJ* addend, const sJ* augend) {
	addend->s1 += augend->s1;
	addend->s4 += augend->s4;
	addend->s5 += augend->s5;
	addend->s6 += augend->s6;
	addend->s4k1 += augend->s4k1;
	addend->s4k3 += augend->s4k3;
	addend->s10k1 += augend->s10k1;
	addend->s10k3 += augend->s10k3;
	addend->s10k5 += augend->s10k5;
	addend->s10k7 += augend->s10k7;
	addend->s10k9 += augend->s10k9;
	if (addend->s4k1 >= 1.0) addend->s4k1 -= (int)addend->s4k1;
	if (addend->s4k3 >= 1.0) addend->s4k3 -= (int)addend->s4k3;
	if (addend->s10k1 >= 1.0) addend->s10k1 -= (int)addend->s10k1;
	if (addend->s10k3 >= 1.0) addend->s10k3 -= (int)addend->s10k3;
	if (addend->s10k5 >= 1.0) addend->s10k5 -= (int)addend->s10k5;
	if (addend->s10k7 >= 1.0) addend->s10k7 -= (int)addend->s10k7;
	if (addend->s10k9 >= 1.0) addend->s10k9 -= (int)addend->s10k9;
}

//not actually quick
__device__ void quickMod(INT_64 input, const INT_64 mod, INT_64 *output) {

	/*INT_64 copy = input;
	INT_64 test = input % mod;*/
	INT_64 temp = mod;
	while (temp < input && !(temp&int64MaxBit)) temp <<= 1;
	if (temp > input) temp >>= 1;
	while (input >= mod && temp >= mod) {
		if (input >= temp) input -= temp;
		temp >>= 1;
	}
	/*if (input != test && !atomicAdd(&printOnce,1))
	{
	printf("input %llu mod %llu error\n", copy, mod);
	printOnce = 1;
	}*/
	*output = input;
}

//binary search to find highest 1 bit in multiplier
__device__ void findMultiplierHighestBit(const TYPE multiplier, TYPE *output) {

	//if no bits are 1 then highest bit doesn't exist
	if (!multiplier) {
		*output = 0;
		return;
	}

	int highestBitLocMax = typeSize;
	int highestBitLocMin = 0;

	int middle = (highestBitLocMax + highestBitLocMin) >> 1;

	TYPE highestBit = 1L;
	highestBit <<= middle;

	int less = highestBit <= multiplier;

	while (!((highestBit << 1) > multiplier && less)) {
		if (less) highestBitLocMin = middle + 1;
		else highestBitLocMax = middle - 1;
		middle = (highestBitLocMax + highestBitLocMin) >> 1;
		//this might not look necessary but it is
		highestBit = 1L;
		highestBit <<= middle;
		less = highestBit <= multiplier;
	}

	/*unsigned long long highestBit2 = 0x8000000000000000;

	while (highestBit2 > multiplier) highestBit2 >>= 1;

	if (highestBit != highestBit2 && !printOnce) {
	printf("multiplier %d error; highestBit %d; highestBit2 %d\n", multiplier, highestBit, highestBit2);
	printOnce = 1;
	}*/

	*output = highestBit;
}

//hacker's delight method to find highest bit in a long long (it just works)
//http://graphics.stanford.edu/~seander/bithacks.html
//just barely faster than built-in CUDA __clzll
__device__ void findMultiplierHighestBitHackersDelight(TYPE multiplier, TYPE *output) {

	multiplier |= multiplier >> 1;
	multiplier |= multiplier >> 2;
	multiplier |= multiplier >> 4;
	multiplier |= multiplier >> 8;
	multiplier |= multiplier >> 16;
	multiplier |= multiplier >> 32;

	*output = multiplier ^ (multiplier >> 1);

}

__device__ void modMultiplyLeftToRight(const TYPE multiplicand, const TYPE multiplier, TYPE mod, TYPE *output) {
	*output = multiplicand;

	TYPE highestBitMask = 0;

	findMultiplierHighestBitHackersDelight(multiplier, &highestBitMask);

	while (highestBitMask > 1) {
		//only perform modulus operation during loop if result is >= (TYPE maximum + 1)/4 (in order to prevent overflowing)
		if (*output >= multiplyModCond) *output %= mod;
		*output <<= 1;
		highestBitMask >>= 1;
		if (multiplier&highestBitMask)	*output += multiplicand;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	*output %= mod;
}

__device__ void modMultiplyRightToLeft(INT_64 multiplicand, INT_64 multiplier, INT_64 mod, INT_64 *output) {
	INT_64 result = 0;

	INT_64 mask = 1;

	while (multiplier > 0) {
		if (multiplier&mask) {
			result += multiplicand;

			//only perform modulus operation during loop if result is >= (INT_64 maximum + 1)/2 (in order to prevent overflowing)
			if (result >= mod) result -= mod;
		}
		multiplicand <<= 1;
		if (multiplicand >= mod) multiplicand -= mod;
		multiplier >>= 1;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//uses 32 bit multiplications to compute the highest 64 and lowest 64 bits of multiplying 2 64 bit numbers together
__device__ void multiply64By64(INT_64 multiplicand, INT_64 multiplier, INT_64 * lo, INT_64 * hi) {

	//a : multiplicand
	//b : multiplier
	//_lo : low 32 bits of result
	//_hi : high 32 bits of result
	asm("{\n\t"
		".reg .u32          t0, t1, t2, t3, v0, v1, v2, v3;\n\t"
		"mov.b64           {v0, v1}, %2;\n\t" //splits a into hi and lo 32 bit words
		"mov.b64           {v2, v3}, %3;\n\t" //splits b into hi and lo 32 bit words
		"mul.lo.u32         t0, v0, v2;    \n\t" //lolo = lo(alo*blo)
		"mul.hi.u32         t1, v0, v2;    \n\t" //lohi = hi(alo*blo)
		"mad.lo.cc.u32      t1, v0, v3, t1;\n\t" //lohi = lo(alo*bhi) + hi(alo*blo) (with carry flag)
		"madc.hi.cc.u32     t2, v0, v3,  0;\n\t" //hilo = hi(alo*bhi) + 1 carry (with carry flag, as carry may need to propagate)
		"madc.hi.u32        t3, v1, v3,  0;\n\t" //hihi = hi(ahi*bhi) + 1 carry (no need to set carry)
		"mad.lo.cc.u32      t1, v1, v2, t1;\n\t" //lohi = lo(ahi*blo) + lo(alo*bhi) + hi(alo*blo) (with carry flag)
		"madc.hi.cc.u32     t2, v1, v2, t2;\n\t" //hilo = hi(ahi*blo) + hi(alo*bhi) + 2 carries (with carry flag)
		"addc.u32           t3, t3, 0;\n\t" //hihi = hi(ahi*bhi) + 2 carries (no need to set carry)
		"mad.lo.cc.u32      t2, v1, v3, t2;\n\t" //hilo = lo(ahi*bhi) + hi(ahi*blo) + hi(alo*bhi) + 2 carries (with carry flag)
		"addc.u32           t3, t3, 0;\n\t" //hihi = hi(ahi*bhi) + 3 carries
		"mov.b64            %0, {t0, t1};\n\t" //concatenates t0 and t1 into 1 64 bit word
		"mov.b64            %1, {t2, t3};\n\t" //concatenates t2 and t3 into 1 64 bit word
		"}"
		: "=l"(*lo), "=l"(*hi)
		: "l"(multiplicand), "l"(multiplier));
}

//uses 32 bit multiplications to compute the highest 64 and lowest 64 bits of multiplying 2 64 bit numbers together
//adds the results to the contents of lo
//basically a 128 bit mad with 64 bit inputs
__device__ void multiply64By64PlusLo(INT_64 multiplicand, INT_64 multiplier, INT_64 * lo, INT_64 * hi) {
	
	//a : multiplicand
	//b : multiplier
	//_lo : low 32 bits of result
	//_hi : high 32 bits of result
	asm("{\n\t"
		".reg .u32          t0, t1, t2, t3, v0, v1, v2, v3;\n\t"
		"mov.b64           {t0, t1}, %4;\n\t" //splits lo into t0 and t1
		"mov.b64           {v0, v1}, %2;\n\t" //splits a into hi and lo 32 bit words
		"mov.b64           {v2, v3}, %3;\n\t" //splits b into hi and lo 32 bit words
		"mad.lo.cc.u32      t0, v0, v2, t0;\n\t" //lolo = starting_value + lo(alo*blo) (with carry flag)
		"madc.hi.cc.u32     t1, v0, v2, t1;\n\t" //lohi = starting_value + hi(alo*blo) + 1 carry (with carry flag)
		"madc.hi.cc.u32     t2, v0, v3,  0;\n\t" //hilo = hi(alo*bhi) + 1 carry (with carry flag, as carry may need to propagate)
		"madc.hi.u32        t3, v1, v3,  0;\n\t" //hihi = hi(ahi*bhi) + 1 carry (no need to set carry)
		"mad.lo.cc.u32      t1, v0, v3, t1;\n\t" //lohi = starting_value + lo(alo*bhi) + hi(alo*blo) + 1 carry (with carry flag)
		"madc.hi.cc.u32     t2, v1, v2, t2;\n\t" //hilo = hi(ahi*blo) + hi(alo*bhi) + 2 carries (with carry flag)
		"addc.u32           t3, t3, 0;\n\t" //hihi = hi(ahi*bhi) + 2 carries (no need to set carry)
		"mad.lo.cc.u32      t1, v1, v2, t1;\n\t" //lohi = starting_value + lo(ahi*blo) + lo(alo*bhi) + hi(alo*blo) + 1 carry (with carry flag)
		"madc.lo.cc.u32     t2, v1, v3, t2;\n\t" //hilo = lo(ahi*bhi) + hi(ahi*blo) + hi(alo*bhi) + 3 carries (with carry flag)
		"addc.u32           t3, t3, 0;\n\t" //hihi = hi(ahi*bhi) + 3 carries
		"mov.b64            %0, {t0, t1};\n\t" //concatenates t0 and t1 into 1 64 bit word
		"mov.b64            %1, {t2, t3};\n\t" //concatenates t2 and t3 into 1 64 bit word
		"}"
		: "=l"(*lo), "=l"(*hi)
		: "l"(multiplicand), "l"(multiplier), "l"(*lo));
}

//uses 32 bit multiplications to compute the highest 64 and lowest 64 bits of multiplying a 32 and 64 bit number together
//adds the results to the contents of lo
__device__ void multiply32By64PlusLo(INT_64 multiplicand, INT_64 multiplier, INT_64 * lo, INT_64 * hi) {

	//a : multiplicand
	//b : multiplier
	//_lo : low 32 bits of result
	//_hi : high 32 bits of result
	asm("{\n\t"
		".reg .u32          t0, t1, t2, t3, v0, v1, v2, v3;\n\t"
		"mov.b64           {t0, t1}, %4;\n\t" //splits lo into t0 and t1
		"mov.b64           {v0, v1}, %2;\n\t" //splits a into hi and lo 32 bit words (although a has no high bits set, we just won't use v1)
		"mov.b64           {v2, v3}, %3;\n\t" //splits b into hi and lo 32 bit words
		"mad.lo.cc.u32      t0, v0, v2, t0;\n\t" //lolo = starting_value + lo(alo*blo) (with carry flag)
		"madc.hi.cc.u32     t1, v0, v2, t1;\n\t" //lohi = starting_value + hi(alo*blo) + 1 carry (with carry flag)
		"madc.hi.cc.u32     t2, v0, v3,  0;\n\t" //hilo = hi(alo*bhi) + 1 carry (with carry flag, as carry may need to propagate)
		"mad.lo.cc.u32      t1, v0, v3, t1;\n\t" //lohi = starting_value + lo(alo*bhi) + hi(alo*blo) + 1 carry (with carry flag)
		"addc.cc.u32           t2, t2, 0;\n\t" //hilo = hi(alo*bhi) + 2 carries (with carry flag, as carry may need to propagate)
		"addc.u32           t3, 0, 0;\n\t" //just incase the last line produced a carry
		"mov.b64            %0, {t0, t1};\n\t" //concatenates t0 and t1 into 1 64 bit word
		"mov.b64            %1, {t2, t3};\n\t" //concatenates t2 and t3 into 1 64 bit word
		"}"
		: "=l"(*lo), "=l"(*hi)
		: "l"(multiplicand), "l"(multiplier), "l"(*lo));
}

//adds augend to addend
//if an overflow is detected, add maxMod to augend
//if that overflows, add it again (as long as the mod for which maxMod is defined is < 2^63, this can't overflow)
//this function allows the program to avoid calculating any modulus operations in modMultiply64Bit except once at the end
//doing this saves anywhere from 25-40% of runtime (with larger savings coming from larger digit calculations)
__device__ void addWithCarryConvertedToMod(INT_64 & addend, const INT_64 & augend, const INT_64 & maxMod) {
	asm("{\n\t"
		".reg .u32         t0;\n\t"
		".reg .pred         %p;\n\t"
		"add.cc.u64        %0, %0, %1;\n\t" //addend = addend + augend
		"addc.u32          t0, 0, 0;\n\t"
		"setp.eq.u32       %p, 1, t0;\n\t"
		"@%p add.cc.u64   %0, %0, %2;\n\t" //if carry-flag set, addend = addend + augend + maxMod - 2^64
		"addc.u32          t0, 0, 0;\n\t"
		"setp.eq.u32       %p, 1, t0;\n\t"
		"@%p add.cc.u64   %0, %0, %2;\n\t" //if carry-flag set, addend = addend + augend + 2*maxMod - 2^65
		"}"
		: "=l"(addend)
		: "l"(augend), "l"(maxMod));
}

__device__ void multiplyAdd64Hi(const INT_64 & multiplicand, const INT_64 & multiplier, INT_64 * accumulate) {
	asm("{\n\t"
		"mad.hi.u64          %0, %1, %2, %3;\n\t"
		"}"
		: "=l"(*accumulate)
		: "l"(multiplicand), "l"(multiplier), "l"(*accumulate));
}

//calculates the 128 bit product of multiplicand and multiplier
//takes the highest 64 bits and multiplies it by maxMod (2^64 % mod) and adds it to the low 64 bits, repeating until the highest 64 bits are zero
//this takes (log2(mod)) / (64 - log2(mod)) steps
//maxMod is constant with respect to each mod, therefore best place to calculate is in modExp functions
__device__ void modMultiply64Bit(INT_64 multiplicand, INT_64 multiplier, const INT_64 & mod, const INT_64 & maxMod, INT_64 & output) {
	INT_64 hi = 0, result = 0;// , lo;
	multiply64By64PlusLo(multiplicand, multiplier, &result, &hi);
	while (hi) {
		if(hi > 0xFFFFFFFF) multiply64By64PlusLo(hi, maxMod, &result, &hi);
		else multiply32By64PlusLo(hi, maxMod, &result, &hi);
	}
	if(result >= mod) result %= mod;
	output = result;
}

//an experiment to see if reducing the number of arguments saves any time
__device__ void modSquare64Bit(INT_64 *number, INT_64 mod, INT_64 maxMod) {
	INT_64	hi = __umul64hi(*number, *number);
	*number = (*number * *number) % mod;
	while (hi) {
		INT_64 lo = hi * maxMod;

		//multiplyModCond should be (2^64)/(number of loop iterations)
		//where loop iterations are roughly 64/(64 - log2(mod))
		//THEREFORE THIS SHOULD NOT BE A COMPILE TIME CONSTANT
		//but a runtime variable set at launch based upon the maximum mod that will be passed to this function
		//for 2^40 number of loops is 2, for 2^50 number of loops is 4
		if (lo > multiplyModCond) lo %= mod;

		*number += lo;
		hi = __umul64hi(hi, maxMod);
	}
	if(*number >= mod) *number %= mod;
}

//leverages a machine instruction that returns the highest 64 bits of the multiplication operation
//multiplicand and multiplier should always be less than mod (may work correctly even if this is not the case)
//uses bitshifts and subtraction to avoid multiplications and modulus respectively inside the loop
//loops more times than other version
//slower than other version (but could be faster if mod is close to 2^63)
__device__ void modMultiply64BitAlt(INT_64 multiplicand, INT_64 multiplier, INT_64 mod, const int modMaxBitPos, INT_64 *output) {
	INT_64	hi = __umul64hi(multiplicand, multiplier);
	INT_64 result = (multiplicand * multiplier) % mod;
	int count = 64;
	while (count > 0) {

		//determine the number of bits to shift hi so that 2*mod > hi > mod
		int dif = __clzll(hi) - modMaxBitPos;

		if (dif > count) dif = count;

		//hi is the highest 64 bits of multiplicand*multiplier
		//so hi is actually hi*2^64
		//takes bits from 2^64 and gives them to hi until 2^64 is reduced to 2^0
		//each step of loop only gives as many bits to hi as to satisfy 2*mod > hi > mod
		hi <<= dif;

		if(hi >= mod) hi -= mod;

		count -= dif;
	}
	*output = (result + hi);
	if(*output > mod) *output -= mod;
}

//perform right-to-left binary exponention taking modulus of both base and result at each step
//64 bit integers are required to accurately find the modular exponents of numbers when mod is >= ~10e6
//however, with CUDA 64 bit integers are implemented at compile time as two 32 bit integers
//this produces about a 10x slowdown over computations using 32 bit integers
__device__ void modExp(unsigned long long base, long exp, long mod, long *output) {
	const unsigned long mask = 1;
	unsigned long long result = 1;

	//only perform modulus operation during loop if result or base is >= 2^32 (in order to prevent either from overflowing)
	//this saves 30% computation time over performing modulus in every loop iteration
	const unsigned long long modCond = 0x100000000;//2^32

	while (exp > 0) {
		if (exp&mask) {
			result *= base;
			if (result >= modCond) result %= mod;
		}
		base *= base;
		if (base >= modCond) base %= mod;
		exp >>= 1;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//greatest common denominator method pulled unmodified from http://www.hackersdelight.org/hdcodetxt/mont64.c.txt

/* C program implementing the extended binary GCD algorithm. C.f.
http://www.ucl.ac.uk/~ucahcjm/combopt/ext_gcd_python_programs.pdf. This
is a modification of that routine in that we find s and t s.t.
gcd(a, b) = s*a - t*b,
rather than the same expression except with a + sign.
This routine has been greatly simplified to take advantage of the
facts that in the MM use, argument a is a power of 2, and b is odd. Thus
there are no common powers of 2 to eliminate in the beginning. The
parent routine has two loops. The first drives down argument a until it
is 1, modifying u and v in the process. The second loop modifies s and
t, but because a = 1 on entry to the second loop, it can be easily seen
that the second loop doesn't alter u or v. Hence the result we want is u
and v from the end of the first loop, and we can delete the second loop.
The intermediate and final results are always > 0, so there is no
trouble with negative quantities. Must have a either 0 or a power of 2
<= 2**63. A value of 0 for a is treated as 2**64. b can be any 64-bit
value.
Parameter a is half what it "should" be. In other words, this function
does not find u and v st. u*a - v*b = 1, but rather u*(2a) - v*b = 1. */

__device__ void xbinGCD(INT_64 a, INT_64 b, INT_64 *pu, INT_64 *pv)
{
	INT_64 alpha, beta, u, v;
	//printf("Doing GCD(%llx, %llx)\n", a, b);

	u = 1; v = 0;
	alpha = a; beta = b;         // Note that alpha is
								 // even and beta is odd.

								 /* The invariant maintained from here on is:
								 2a = u*2*alpha - v*beta. */

								 // printf("Before, a u v = %016llx %016llx %016llx\n", a, u, v);
	while (a > 0) {
		a = a >> 1;
		if ((u & 1) == 0) {             // Delete a common
			u = u >> 1; v = v >> 1;      // factor of 2 in
		}                               // u and v.
		else {
			/* We want to set u = (u + beta) >> 1, but
			that can overflow, so we use Dietz's method. */
			u = ((u ^ beta) >> 1) + (u & beta);
			v = (v >> 1) + alpha;
		}
		//    printf("After,  a u v = %016llx %016llx %016llx\n", a, u, v);
	}

	// printf("At end,    a u v = %016llx %016llx %016llx\n", a, u, v);
	*pu = u;
	*pv = v;
	return;
}


//montgomery multiplication method from http://www.hackersdelight.org/hdcodetxt/mont64.c.txt
//slightly modified to use more efficient 64 bit multiply-adds in PTX assembly
__device__ void montgomeryMult(INT_64 abar, INT_64 bbar, INT_64 mod, INT_64 mprime, INT_64 & output) {

	INT_64 thi = 0, tlo = 0, tm = 0, tmmhi = 0, tmmlo = 0, uhi = 0, ulo = 0, ov = 0;

	//printf("\nmontmul, abar = %016llx, bbar   = %016llx\n", abar, bbar);
	//printf("            m = %016llx, mprime = %016llx\n", m, mprime);

	/* t = abar*bbar. */

	multiply64By64(abar, bbar, &tlo, &thi);

	//unless tlo is zero here, there will always be a carry from tm*mod + tlo
	INT_64 lowerCarry = (tlo > 0);

	//this would only be a problem if thi was 2^64 - 1
	//which can never occur if mod is representable in an unsigned long long
	thi += lowerCarry;

	/* Now compute u = (t + ((t*mprime) & mask)*m) >> 64.
	The mask is fixed at 2**64-1. Because it is a 64-bit
	quantity, it suffices to compute the low-order 64
	bits of t*mprime, which means we can ignore thi. */

	tm = tlo * mprime;
	
	//there is an optimization to be made here, tm = lo64(tlo * mprime)
	//so tm * mod = lo64(tlo * mprime) * mod
	//but mprime*mod is constant for a given mod
	//is there a way to reduce the amount of work from this?
	//multiply64By64PlusLo(tm, mod, &tlo, &tmmhi);
	multiply64By64(tm, mod, &tlo, &tmmhi);//tlo is not used
	uhi = thi + tmmhi;

	// The above addition can overflow. Detect that here.
	//tmmhi will only be zero if tlo was zero above
	//so an overflow can only exist with a non-zero tmmhi
	//also if mod is < 2^63 this can't overflow, so no need to check
	//ov = (uhi < thi);
	// if (ov > 0 || ulo >= mod)      // If u >= mod,
	//    ulo = ulo - mod;            // subtract mod from u.
	//uhi = uhi - (mod & -(ov | (uhi >= mod))); // Alternative
										  // with no branching.
	
	//assumes mod < 2^63, WILL NOT WORK if mod > 2^63 because overflow can exist in above addition in that case
	//if (uhi >= mod) uhi -= mod;
	//in addition to mitigating most GPUs' poor conditional branching performance, unconditional code execution is also resistant to side-channel attacks
	uhi = uhi - (mod & -((uhi >= mod)));

	output = uhi;
}

//using left-to-right binary exponentiation
//the position of the highest bit in exponent is passed into the function as a parameter (it is more efficient to find it outside)
//uses montgomery multiplication to reduce difficulty of modular multiplication (runs in 55% of runtime of non-montgomery modular multiplication)
//montgomery multiplication suggested by njuffa
__device__ void modExpLeftToRight(const INT_64 & exp, const INT_64 & mod, INT_64 highestBitMask, INT_64 & output) {

	if (!exp) {
		//no need to set output to anything as it is already 1
		return;
	}

	INT_64 rInverse, mPrime;

	//finds rInverse*2^64 - mPrime*mod = 1
	xbinGCD(int64MaxBit, mod, &rInverse, &mPrime);

	INT_64 result;

	INT_64 maxMod = int64MaxBit % mod;

	maxMod <<= 1;
	
	if (maxMod > mod) maxMod -= mod;

	//baseSystem*2^64 % mod
	modMultiply64Bit(maxMod, baseSystem, mod, maxMod, result);

	//save this to use in loop
	INT_64 baseBar = result;

	while (highestBitMask > 1) {

		montgomeryMult(result, result, mod, mPrime, result);//result^2

		highestBitMask >>= 1;
		if (exp&highestBitMask) montgomeryMult(result, baseBar, mod, mPrime, result);//result*base
	}

	//convert result out of montgomery form
	modMultiply64Bit(result, rInverse, mod, maxMod, result);

	output = result;
}

//find ( baseSystem^n % mod ) / mod and add to partialSum
//experimented with placing forceinline and noinline on various functions again
//with new changes, noinline now has most effect here, no idea why
__device__ __noinline__ void fractionalPartOfSum(const TYPE & exp, const TYPE & mod, double *partialSum, TYPE highestBitMask, const int & negative) {
	TYPE expModResult = 1;
	modExpLeftToRight(exp, mod, highestBitMask, expModResult);
	double sumTerm = (((double)expModResult) / ((double)mod));
	
	//if n is odd, then sumTerm will be negative
	//add 1 to it to find its positive fractional part
	if (negative) sumTerm = 1.0 - sumTerm;
	*partialSum += sumTerm;
	if((*partialSum) > 1.0) *partialSum -= (int)(*partialSum);
}

//stride over all parts of summation in bbp formula where k <= n
//to compute partial sJ sums
__device__ void bbp(TYPE n, TYPE start, INT_64 end, int gridId, TYPE stride, sJ* output, volatile INT_64* progress, int progressCheck) {

	TYPE highestExpBit = 1;
	while (highestExpBit <= n)	highestExpBit <<= 1;
	for (TYPE k = start; k <= end; k += stride) {
		while (highestExpBit > (n - k))  highestExpBit >>= 1;
		TYPE mod = 4 * k + 1;
		fractionalPartOfSum(n - k, mod, &((*output).s4k1), highestExpBit, k & 1);
		mod += 2;//4k + 3
		fractionalPartOfSum(n - k, mod, &((*output).s4k3), highestExpBit, k & 1);
		mod = 10 * k + 1;
		fractionalPartOfSum(n - k, mod, &((*output).s10k1), highestExpBit, k & 1);
		mod += 2;//10k + 3
		fractionalPartOfSum(n - k, mod, &((*output).s10k3), highestExpBit, k & 1);
		mod += 2;//10k + 5
		fractionalPartOfSum(n - k, mod, &((*output).s10k5), highestExpBit, k & 1);
		mod += 2;//10k + 7
		fractionalPartOfSum(n - k, mod, &((*output).s10k7), highestExpBit, k & 1);
		mod += 2;//10k + 9
		fractionalPartOfSum(n - k, mod, &((*output).s10k9), highestExpBit, k & 1);
		if (!progressCheck) {
			//only 1 thread (with gridId 0 on GPU0) ever updates the progress
			*progress = k;
		}
	}
}

//determine from thread and block position where to begin stride
//only one of the threads per kernel (AND ONLY ON GPU0) will report progress
__global__ void bbpKernel(sJ *c, volatile INT_64 *progress, TYPE digit, int gpuNum, INT_64 begin, INT_64 end, INT_64 stride)
{
	int gridId = threadIdx.x + blockDim.x * blockIdx.x;
	TYPE start = begin + gridId + blockDim.x * gridDim.x * gpuNum;
	int progressCheck = gridId + blockDim.x * gridDim.x * gpuNum;
	bbp(digit, start, end, gridId, stride, c + gridId, progress, progressCheck);
}

//stride over current leaves of reduce tree
__global__ void reduceSJKernel(sJ *c, int offset, int stop) {
	int stride = blockDim.x * gridDim.x;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	while (i < stop) {
		int augend = i + offset;
		sJAdd(c + i, c + augend);
		i += stride;
	}
}

//standard tree-based parallel reduce
hipError_t reduceSJ(sJ *c, unsigned int size) {
	hipError_t cudaStatus;
	while (size > 1) {
		int nextSize = (size + 1) >> 1;

		//size is odd
		if (size&1) reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize - 1);
		//size is even
		else reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceSJKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceSJKernel!\n", cudaStatus);
			return cudaStatus;
		}

		size = nextSize;
	}
	return cudaStatus;
}

//compute four steps of sJ sums for i > n and add to sJ sums found previously
//combine sJs according to bbp formula
//multiply by 16^5 to extract five digits of pi starting at n
long finalizeDigit(sJ input, TYPE n) {
	double reducer = 1.0;
	double s1 = input.s1;
	double s4 = input.s4;
	double s5 = input.s5;
	double s6 = input.s6;
	double trash = 0.0;
	if (n < 16000) {
		for (int i = 0; i < 4; i++) {
			n++;
			reducer /= (double)baseSystem;
			double eightN = 8.0 * n;
			s1 += reducer / (eightN + 1.0);
			s4 += reducer / (eightN + 4.0);
			s5 += reducer / (eightN + 5.0);
			s6 += reducer / (eightN + 6.0);
		}
	}
	//remove any integer part of s1-s6
	s1 = modf(s1, &trash);
	s4 = modf(s4, &trash);
	s5 = modf(s5, &trash);
	s6 = modf(s6, &trash);
	double hexDigit = 4.0*s1 - 2.0*s4 - s5 - s6;
	hexDigit = modf(hexDigit, &trash);
	if (hexDigit < 0) hexDigit++;
	hexDigit *= baseSystem*baseSystem*baseSystem*baseSystem*baseSystem;
	printf("hexDigit = %.8f\n", hexDigit);
	return (long)hexDigit;
}

INT_64 finalizeDigitAlt(sJ input, TYPE n) {
	double reducer = 1.0;

	//unfortunately 64 is not a power of 16, so if n is < 2
	//then division is unavoidable
	//this division must occur before any modulus are taken
	if(n == 0) reducer /= 64.0;
	else if (n == 1) reducer /= 4.0;

	//logic relating to 1024 not being a power of 16 and having to divide by 64
	int loopLimit = (2 * n - 3) % 5;
	if (n < 2) n = 0;
	else n = (2 * n - 3) / 5;

	double trash = 0.0;
	double s4k1 = input.s4k1 * reducer;//modf(input.s4k1, &trash);
	double s4k3 = input.s4k3 * reducer;//modf(input.s4k3, &trash);
	double s10k1 = input.s10k1 * reducer;//modf(input.s10k1, &trash);
	double s10k3 = input.s10k3 * reducer;//modf(input.s10k3, &trash);
	double s10k5 = input.s10k5 * reducer;//modf(input.s10k5, &trash);
	double s10k7 = input.s10k7 * reducer;//modf(input.s10k7, &trash);
	double s10k9 = input.s10k9 * reducer;//modf(input.s10k9, &trash);
	
	if (n < 16000) {
		for (int i = 0; i < 5; i++) {
			n++;
			double sign = 1.0;
			double nD = (double)n;
			if (n & 1) sign = -1.0;
			reducer /= (double)baseSystem;
			s4k1 += sign * reducer / (4.0 * nD + 1.0);
			s4k3 += sign * reducer / (4.0 * nD + 3.0);
			s10k1 += sign * reducer / (10.0 * nD + 1.0);
			s10k3 += sign * reducer / (10.0 * nD + 3.0);
			s10k5 += sign * reducer / (10.0 * nD + 5.0);
			s10k7 += sign * reducer / (10.0 * nD + 7.0);
			s10k9 += sign * reducer / (10.0 * nD + 9.0);
		}
	}

	//multiply sJs by coefficients from Bellard's formula and then find their fractional parts
	s4k1 = modf(-32.0*s4k1, &trash);
	if (s4k1 < 0) s4k1++;
	s4k3 = modf(-1.0*s4k3, &trash);
	if (s4k3 < 0) s4k3++;
	s10k1 = modf(256.0*s10k1, &trash);
	if (s10k1 < 0) s10k1++;
	s10k3 = modf(-64.0*s10k3, &trash);
	if (s10k3 < 0) s10k3++;
	s10k5 = modf(-4.0*s10k5, &trash);
	if (s10k5 < 0) s10k5++;
	s10k7 = modf(-4.0*s10k7, &trash);
	if (s10k7 < 0) s10k7++;
	s10k9 = modf(s10k9, &trash);
	if (s10k9 < 0) s10k9++;

	double hexDigit = s4k1 + s4k3 + s10k1 + s10k3 + s10k5 + s10k7 + s10k9;
	hexDigit = modf(hexDigit, &trash);
	if (hexDigit < 0) hexDigit++;

	//16^n is divided by 64 and then combined into chunks of 1024^m
	//where m is = (2n - 3)/5
	//because 5 may not evenly divide this, the remaining 4^((2n - 3)%5)
	//must be multiplied into the formula at the end
	for (int i = 0; i < loopLimit; i++) hexDigit *= 4.0;
	hexDigit = modf(hexDigit, &trash);

	//shift left by 8 hex digits
	for (int i = 0; i < 12; i++) hexDigit *= 16.0;
	printf("hexDigit = %.8f\n", hexDigit);
	return (INT_64)hexDigit;
}

void checkForProgressCache(INT_64 digit, INT_64 * contFrom, sJ * cache, double * previousTime) {
	std::string target = "digit" + std::to_string(digit) + "Base";
	std::string pToFile;
	int found = 0;
	for (auto& element : std::experimental::filesystem::directory_iterator("progressCache")) {
		std::string name = element.path().filename().string();
		//filename begins with desired string
		if (name.compare(0, target.length(), target) == 0) {
			pToFile = element.path().string();
			found = 1;
		}
		else if (found) {
			break;
		}
	}
	if (found) {
		int chosen = 0;
		while (!chosen) {
			chosen = 1;
			std::cout << "A cache of a previous computation for this digit exists." << std::endl;
			std::cout << "Would you like to reload the most recent cache (" << pToFile << ")? y\\n" << std::endl;
			char choice;
			std::cin >> choice;
			if (choice == 'y') {
				std::cout << "Loading cache and continuing computation." << std::endl;
				try {
					std::ifstream file;
					file.open(pToFile);

					file >> *contFrom;

					//theoretically the standard specifies that this works for doubles
					//however msvc doesn't output correctly for doubles with hexfloat (it outputs as a float)
					//but it appears to work correctly for reading into doubles as tested so far
					file >> std::hexfloat >> *previousTime;
					file >> std::hexfloat >> cache->s4k1;
					file >> std::hexfloat >> cache->s4k3;
					file >> std::hexfloat >> cache->s10k1;
					file >> std::hexfloat >> cache->s10k3;
					file >> std::hexfloat >> cache->s10k5;
					file >> std::hexfloat >> cache->s10k7;
					file >> std::hexfloat >> cache->s10k9;
				}
				catch(std::ifstream::failure& e) {
					fprintf(stderr, "Error opening file %s\n", pToFile.c_str());
					fprintf(stderr, "%s\n", e.what());
					std::cout << "Could not reload cache. Beginning computation without reloading." << std::endl;
				}
			}
			else if (choice == 'n') {
				std::cout << "Beginning computation without reloading." << std::endl;
			}
			else {
				std::cout << "Invalid input" << std::endl;
				// Ignore to the end of line
				std::cin.clear();
				std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
				chosen = 0;
			}
		}
	}
	else {
		std::cout << "No progress cache file found. Beginning computation without reloading." << std::endl;
	}
}

int main()
{
	try {
		const int arraySize = threadCountPerBlock * blockCount;
		INT_64 hexDigitPosition;
		std::cout << "Input hexDigit to calculate (1-indexed):" << std::endl;
		std::cin >> hexDigitPosition;
		//subtract 1 to convert to 0-indexed
		hexDigitPosition--;

		INT_64 sumEnd = 0;

		//convert from number of digits in base16 to base1024
		//because of the 1/64 in formula, we must subtract log16(64) which is 1.5, so carrying the 2 * (digitPosition - 1.5) = 2 * digitPosition - 3
		//this is because division messes up with respect to modulus, so use the 16^digitPosition to absorb it
		if (hexDigitPosition < 2) sumEnd = 0;
		else sumEnd = ((2LLU * hexDigitPosition) - 3LLU) / 5LLU;

		INT_64 beginFrom = 0;
		sJ cudaResult;
		double previousTime = 0.0;
		checkForProgressCache(sumEnd, &beginFrom, &cudaResult, &previousTime);

		HANDLE handles[totalGpus];
		BBPLAUNCHERDATA gpuData[totalGpus];

		clock_t start = clock();

		PPROGRESSDATA prog = setupProgress();

		if (prog->error != hipSuccess) return 1;
		prog->begin = start;
		prog->maxProgress = sumEnd;
		prog->previousCache = cudaResult;
		prog->previousTime = previousTime;

		HANDLE progThread = CreateThread(NULL, 0, *progressCheck, (LPVOID)prog, 0, NULL);

		if (progThread == NULL) {
			fprintf(stderr, "progressCheck thread creation failed\n");
			return 1;
		}

		for (int i = 0; i < totalGpus; i++) {

			gpuData[i].digit = sumEnd;
			gpuData[i].gpu = i;
			gpuData[i].totalGpus = totalGpus;
			gpuData[i].size = arraySize;
			gpuData[i].deviceProg = prog->deviceProg;
			gpuData[i].status = &(prog->status[i]);
			gpuData[i].dataWritten = &(prog->dataWritten);
			gpuData[i].nextStrideBegin = &(prog->nextStrideBegin[i]);
			gpuData[i].beginFrom = beginFrom;


			handles[i] = CreateThread(NULL, 0, *cudaBbpLauncher, (LPVOID)&(gpuData[i]), 0, NULL);

			if (handles[i] == NULL) {
				fprintf(stderr, "gpu%dThread failed to launch\n", i);
				return 1;
			}
		}

		hipError_t cudaStatus;

		for (int i = 0; i < totalGpus; i++) {

			WaitForSingleObject(handles[i], INFINITE);
			CloseHandle(handles[i]);

			cudaStatus = gpuData[i].error;
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "cudaBbpLaunch failed on gpu%d!\n", i);
				return 1;
			}

			sJ output = gpuData[i].output;

			//sum results from gpus
			sJAdd(&cudaResult, &output);
		}

		//tell the progress thread to quit
		prog->quit = 1;

		WaitForSingleObject(progThread, INFINITE);
		CloseHandle(progThread);

		free(prog);

		INT_64 hexDigit = finalizeDigitAlt(cudaResult, hexDigitPosition);

		clock_t end = clock();

		printf("pi at hexadecimal digit %llu is %012llX\n",
			hexDigitPosition + 1, hexDigit);

		printf("Computed in %.8f seconds\n", previousTime + ((double)(end - start) / (double) CLOCKS_PER_SEC));

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
		}

		return 0;
	}
	catch(...) {
		printf("oops xD\n");
		return 1;
	}
}

PPROGRESSDATA setupProgress() {
	PPROGRESSDATA threadData = new PROGRESSDATA();

	std::atomic_init(&threadData->dataWritten, 0);

	//these variables are linked between host and device memory allowing each to communicate about progress
	volatile INT_64 *currProgHost, *currProgDevice;

	//allow device to map host memory for progress ticker
	threadData->error = hipSetDeviceFlags(hipDeviceMapHost);
	if (threadData->error != hipSuccess) {
		fprintf(stderr, "hipSetDeviceFlags failed with error: %s\n", hipGetErrorString(threadData->error));
		return threadData;
	}

	// Allocate Host memory for progress ticker
	threadData->error = hipHostAlloc((void**)&currProgHost, sizeof(INT_64), hipHostMallocMapped);
	if (threadData->error != hipSuccess) {
		fprintf(stderr, "cudaHostAalloc failed!");
		return threadData;
	}

	//create link between between host and device memory for progress ticker
	threadData->error = hipHostGetDevicePointer((INT_64 **)&currProgDevice, (INT_64 *)currProgHost, 0);
	if (threadData->error != hipSuccess) {
		fprintf(stderr, "hipHostGetDevicePointer failed!");
		return threadData;
	}

	*currProgHost = 0;

	threadData->deviceProg = currProgDevice;
	threadData->currentProgress = currProgHost;
	threadData->quit = 0;

	return threadData;
}

//this function is meant to be run by an independent thread to output progress to the console
DWORD WINAPI progressCheck(LPVOID data) {
	PPROGRESSDATA progP = (PPROGRESSDATA)data;

	double lastProgress = 0;

	std::deque<double> progressQ;
	int count = 0;
	while(!progP->quit) {
		count++;
		double progress = (double)(*(progP->currentProgress)) / (double)progP->maxProgress;

		progressQ.push_front(progress - lastProgress);

		if (progressQ.size() > 100) progressQ.pop_back();

		double progressAvg = 0.0;

		for (std::deque<double>::iterator it = progressQ.begin(); it != progressQ.end(); *it++) progressAvg += *it;

		progressAvg /= (double) progressQ.size();

		double timeEst = 0.01*(1.0 - progress) / (progressAvg);
		lastProgress = progress;
		double time = progP->previousTime + ((double)(clock() - progP->begin) / (double)CLOCKS_PER_SEC);
		//only print every 10th cycle or 0.1 seconds
		if (count == 10) {
			count = 0;
			printf("Current progress is %3.3f%%. Estimated total runtime remaining is %8.3f seconds. Avg rate is %1.5f%%. Time elapsed is %8.3f seconds.\n", 100.0*progress, timeEst, progressAvg*10000.0, time);
		}

		int expected = totalGpus;

		if (std::atomic_compare_exchange_strong(&progP->dataWritten, &expected, 0)) {

			//ensure all sJs in status are from same stride
			//this should always be the case since each 1000 strides are separated by about 90 seconds currently
			//it would be very unlikely for one gpu to get 1000 strides ahead of another, unless the GPUs were not the same
			int sJsAligned = 1;
			INT_64 contProcess = progP->nextStrideBegin[0];
			for (int i = 1; i < totalGpus; i++) sJsAligned &= (progP->nextStrideBegin[i] == contProcess);
			
			if (sJsAligned) {

				char buffer[100];

				double savedProgress = (double) (contProcess - 1LLU) / (double)progP->maxProgress;

				snprintf(buffer, sizeof(buffer), "progressCache/digit%lluBase1024Progress%09.6f.dat", progP->maxProgress, 100.0*savedProgress);

				//would like to do this with ofstream and std::hexfloat
				//but msvc is a microsoft product so...
				FILE * file;
				file = fopen(buffer, "w+");
				if(file != NULL) {
					printf("Writing data to disk\n");
					fprintf(file,"%llu\n",contProcess);
					fprintf(file, "%a\n", time);
					sJ currStatus = progP->previousCache;
					for (int i = 0; i < totalGpus; i++) {
						sJAdd(&currStatus, progP->status + i);
					}
					fprintf(file, "%a\n", currStatus.s4k1);
					fprintf(file, "%a\n", currStatus.s4k3);
					fprintf(file, "%a\n", currStatus.s10k1);
					fprintf(file, "%a\n", currStatus.s10k3);
					fprintf(file, "%a\n", currStatus.s10k5);
					fprintf(file, "%a\n", currStatus.s10k7);
					fprintf(file, "%a", currStatus.s10k9);
					fclose(file);
				}
				else {
					fprintf(stderr, "Error opening file %s\n", buffer);
				}
			}
			else {
				fprintf(stderr, "sJs are misaligned, could not write to disk!\n");
				for (int i = 0; i < totalGpus; i++) fprintf(stderr, "sJ %d alignment is %llu\n", i, progP->nextStrideBegin[i]);
			}
		}

		Sleep(10);
	}
	return 0;
}

// Helper function for using CUDA
DWORD WINAPI cudaBbpLauncher(LPVOID dataV)//hipError_t addWithCuda(sJ *output, unsigned int size, TYPE digit)
{
	PBBPLAUNCHERDATA data = (PBBPLAUNCHERDATA)dataV;
	int size = data->size;
	int gpu = data->gpu;
	int totalGpus = data->totalGpus;
	INT_64 digit = data->digit;
	volatile INT_64 * currProgDevice = data->deviceProg;
	sJ *dev_c = 0;
	sJ* c = new sJ[1];
	sJ *dev_ex = 0;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(gpu);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffer for temp vector
	cudaStatus = hipMalloc((void**)&dev_ex, size * sizeof(sJ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffer for output vector
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(sJ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	INT_64 stride =  (INT_64) size * (INT_64) totalGpus;

	INT_64 launchWidth = stride * 64LLU;

	//need to round up
	//because bbp condition for stopping is <= digit, number of total elements in summation is 1 + digit
	//even when digit/launchWidth is an integer, it is necessary to add 1
	INT_64 neededLaunches = ((digit - data->beginFrom) / launchWidth) + 1LLU;

	for (INT_64 launch = 0; launch < neededLaunches; launch++) {

		INT_64 begin = data->beginFrom + (launchWidth * launch);
		INT_64 end = data->beginFrom + (launchWidth * (launch + 1)) - 1;
		if (end > digit) end = digit;

		// Launch a kernel on the GPU with one thread for each element.
		bbpKernel << <blockCount, threadCountPerBlock >> > (dev_c, currProgDevice, digit, gpu, begin, end, stride);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "bbpKernel launch failed on gpu%d: %s\n", gpu, hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bbpKernel on gpu 1!\n", cudaStatus);
			goto Error;
		}

		//on every 1000th launch write data to status buffer for progress thread to save
		if (launch % 1000 == 0 && launch) {

			//copy current results into temp array to reduce and update status
			cudaStatus = hipMemcpy(dev_ex, dev_c, size * sizeof(sJ), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed in status update!\n");
				goto Error;
			}

			cudaStatus = reduceSJ(dev_ex, size);

			if (cudaStatus != hipSuccess) {
				goto Error;
			}

			// Copy result (reduced into first element) from GPU buffer to host memory.
			cudaStatus = hipMemcpy(c, dev_ex, 1 * sizeof(sJ), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed in status update!\n");
				goto Error;
			}

			*(data->status) = c[0];
			*(data->nextStrideBegin) = data->beginFrom + (launchWidth * (launch + 1LLU));
			std::atomic_fetch_add(data->dataWritten, 1);
		}

		//give the rest of the computer some gpu time to reduce system choppiness
		Sleep(1);
	}

	cudaStatus = reduceSJ(dev_c, size);

	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	// Copy result (reduced into first element) from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, 1 * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	(*data).output = c[0];

Error:
	free(c);
	hipFree(dev_c);
	hipFree(dev_ex);

	(*data).error = cudaStatus;
	return 0;
}
