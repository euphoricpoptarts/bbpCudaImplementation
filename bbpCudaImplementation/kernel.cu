#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>
#include <math.h>
#include <chrono>
#include <thread>
#include <deque>
#include <mutex>
#include <atomic>
#include <iostream>
#ifdef __linux__
#include <experimental/filesystem>
#elif _WIN64
#include <filesystem>
#endif
#include <string>
#include <algorithm>

#define uint32 unsigned int
#define uint64 unsigned long long
#define fastModLimit 0xffffff
//#define QUINTILLION

namespace chr = std::chrono;

std::string propertiesFile = "application.properties";
int totalGpus;
uint64 strideMultiplier;
//warpsize is 32 so optimal value is almost certainly a multiple of 32
const int threadCountPerBlock = 128;
//blockCount is trickier, and is probably a multiple of the number of streaming multiprocessors in a given gpu
int blockCount;
__device__  __constant__ const uint64 twoTo63Power = 0x8000000000000000;
__device__ int printOnce = 0;
int primaryGpu;
int benchmarkBlockCounts;
int numRuns;
uint64 benchmarkTarget;
int startBlocks, blocksIncrement, incrementLimit;
const uint64 cachePeriod = 20000;

struct sJ {
	uint64 s[2] = { 0, 0};
};

__global__ void bbpKernel(sJ *c, uint64 *progress, uint64 startingExponent, uint64 begin, uint64 end, uint64 stride);
hipError_t reduceSJ(sJ *c, unsigned int size);

//adds all elements of addend and augend, storing in addend
__device__ __host__ void sJAdd(sJ* addend, const sJ* augend) {
	addend->s[0] += augend->s[0];
	addend->s[1] += augend->s[1];
	if (addend->s[0] < augend->s[0]) addend->s[1]++;
}

class digitData {
public:
	uint64 sumEnd = 0;
	uint64 startingExponent = 0;
	uint64 beginFrom = 0;

	digitData(uint64 digitInput) {
		//subtract 1 to convert to 0-indexed
		digitInput--;

		//4*hexDigitPosition converts from exponent of 16 to exponent of 2
		//adding 128 for fixed-point division algorithm
		//adding 8 for maximum size of coefficient (so that all coefficients can be expressed by subtracting an integer from the exponent)
		//subtracting 6 for the division by 64 of the whole sum
		this->startingExponent = (4LLU * digitInput) + 128LLU + 2LLU;

		//the end of the sum does not have the addition by 8 so that all calculations will be a positive exponent of 2 after factoring in the coefficient
		//this leaves out a couple potentially positive exponents of 2 (could potentially just check subtraction in modExpLeftToRight and keep the addition by 8)
		this->sumEnd = (4LLU * digitInput - 6LLU + 128LLU) / 10LLU;
	}
};

class progressData {
public:
	volatile uint64 * currentProgress;
	uint64 * deviceProg;
	sJ previousCache;
	double previousTime;
	std::deque<std::pair<sJ, uint64>> * currentResult;
	digitData * digit;
	volatile int quit = 0;
	hipError_t error;
	chr::high_resolution_clock::time_point * begin;
	std::mutex * queueMtx;
	std::atomic<uint64> launchCount;

	progressData(int gpus) {
		std::atomic_init(&this->launchCount, 0);

		//these variables are linked between host and device memory allowing each to communicate about progress
		volatile uint64 *currProgHost;
		uint64 * currProgDevice;

		this->currentResult = new std::deque<std::pair<sJ, uint64>>[gpus];
		this->queueMtx = new std::mutex[gpus];

		//allow device to map host memory for progress ticker
		this->error = hipSetDeviceFlags(hipDeviceMapHost);
		if (this->error != hipSuccess) {
			fprintf(stderr, "hipSetDeviceFlags failed with error: %s\n", hipGetErrorString(this->error));
			return;
		}

		// Allocate Host memory for progress ticker
		this->error = hipHostAlloc((void**)&currProgHost, sizeof(uint64), hipHostMallocMapped);
		if (this->error != hipSuccess) {
			fprintf(stderr, "cudaHostAalloc failed!");
			return;
		}

		//create link between between host and device memory for progress ticker
		this->error = hipHostGetDevicePointer((uint64 **)&currProgDevice, (uint64 *)currProgHost, 0);
		if (this->error != hipSuccess) {
			fprintf(stderr, "hipHostGetDevicePointer failed!");
			return;
		}

		*currProgHost = 0;

		this->deviceProg = currProgDevice;
		this->currentProgress = currProgHost;
		this->quit = 0;
	}

	~progressData() {
		delete[] this->currentResult;
		delete[] this->queueMtx;
		//TODO: delete the device/host pointers?
	}

	int checkForProgressCache(digitData * data) {
		this->digit = data;
		std::string target = "exponent" + std::to_string(this->digit->startingExponent) + "Base";
		std::string pToFile;
		std::vector<std::string> matching;
		int found = 0;
		for (auto& element : std::experimental::filesystem::directory_iterator("progressCache")) {
			std::string name = element.path().filename().string();
			//filename begins with desired string
			if (name.compare(0, target.length(), target) == 0) {
				matching.push_back(element.path().string());
				found = 1;
			}
		}
		if (found) {
			//sort and choose alphabetically last result
			std::sort(matching.begin(), matching.end());
			pToFile = matching.back();

			int chosen = 0;
			while (!chosen) {
				chosen = 1;
				std::cout << "A cache of a previous computation for this digit exists." << std::endl;
				std::cout << "Would you like to reload the most recent cache (" << pToFile << ")? y\\n" << std::endl;
				char choice;
				std::cin >> choice;
				if (choice == 'y') {
					std::cout << "Loading cache and continuing computation." << std::endl;
					FILE * cacheF = fopen(pToFile.c_str(), "r");

					if (cacheF == NULL) {
						std::cout << "Could not open " << pToFile << "!" << std::endl;
						return 1;
					}

					int readLines = 0;

					readLines += fscanf(cacheF, "%llu", &this->digit->beginFrom);
					readLines += fscanf(cacheF, "%la", &this->previousTime);
					for (int i = 0; i < 2; i++) readLines += fscanf(cacheF, "%llX", &this->previousCache.s[i]);
					fclose(cacheF);
					//4 lines of data should have been read, 1 continuation point, 1 time, and 2 data points
					if (readLines != 4) {
						std::cout << "Data reading failed!" << std::endl;
						return 1;
					}
				}
				else if (choice == 'n') {
					std::cout << "Beginning computation without reloading." << std::endl;
				}
				else {
					std::cout << "Invalid input" << std::endl;
					// Ignore to the end of line
					std::cin.clear();
					std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
					chosen = 0;
				}
			}
		}
		else {
			std::cout << "No progress cache file found. Beginning computation without reloading." << std::endl;
		}
		return 0;
	}

	//this function is meant to be run by an independent thread to output progress to the console
	void progressCheck() {

		std::deque<double> progressQ;
		std::deque<chr::high_resolution_clock::time_point> timeQ;
		int count = 0;
		while (!this->quit) {
			count++;
			double progress = (double)(*(this->currentProgress)) / (double)this->digit->sumEnd;

			chr::high_resolution_clock::time_point now = chr::high_resolution_clock::now();
			progressQ.push_front(progress);
			timeQ.push_front(now);

			//progressQ and timeQ should be same size at all times
			if (progressQ.size() > 100) {
				progressQ.pop_back();
				timeQ.pop_back();
			}

			double progressInPeriod = progressQ.front() - progressQ.back();
			double elapsedPeriod = chr::duration_cast<chr::duration<double>>(timeQ.front() - timeQ.back()).count();
			double progressPerSecond = progressInPeriod / elapsedPeriod;

			double timeEst = (1.0 - progress) / (progressPerSecond);
			//find time elapsed during runtime of program, and add it to recorded runtime of previous unfinished run
			double time = this->previousTime + (chr::duration_cast<chr::duration<double>>(now - *this->begin)).count();
			//only print every 10th cycle or 0.1 seconds
			if (count == 10) {
				count = 0;
				printf("Current progress is %3.3f%%. Estimated total runtime remaining is %8.3f seconds. Avg rate is %1.5f%%. Time elapsed is %8.3f seconds.\n", 100.0*progress, timeEst, 100.0*progressPerSecond, time);
			}

			bool resultsReady = true;

			for (int i = 0; i < totalGpus; i++) resultsReady = resultsReady && (this->currentResult[i].size() > 0);

			if (resultsReady) {

				uint64 contProcess = this->currentResult[0].front().second;

				char buffer[100];

				double savedProgress = (double)(contProcess - 1LLU) / (double)this->digit->sumEnd;

				snprintf(buffer, sizeof(buffer), "progressCache/exponent%lluBase2Progress%09.6f.dat", this->digit->startingExponent, 100.0*savedProgress);

				//would like to do this with ofstream and std::hexfloat
				//but msvc is a microsoft product so...
				FILE * file;
				file = fopen(buffer, "w+");
				if (file != NULL) {
					printf("Writing data to disk\n");
					fprintf(file, "%llu\n", contProcess);
					fprintf(file, "%a\n", time);
					sJ currStatus = this->previousCache;
					for (int i = 0; i < totalGpus; i++) {
						this->queueMtx[i].lock();
						sJAdd(&currStatus, &this->currentResult[i].front().first);
						this->currentResult[i].pop_front();
						this->queueMtx[i].unlock();
					}
					for (int i = 0; i < 2; i++) fprintf(file, "%llX\n", currStatus.s[i]);
					fclose(file);
				}
				else {
					fprintf(stderr, "Error opening file %s\n", buffer);
				}
			}

			std::this_thread::sleep_for(std::chrono::milliseconds(10));
		}
	}
};

class bbpLauncher {
public:
	static int totalLaunchers;
	sJ output;
	int gpu = 0;
	int totalGpus = 0;
	int size = 0;
	hipError_t error;
	digitData * data;
	progressData * prog;

	bbpLauncher() {
		this->gpu = totalLaunchers++;
	}

	void initialize(digitData * data, progressData * prog) {
		this->data = data;
		this->prog = prog;
	}

	// Helper function for using CUDA
	void launch()//hipError_t addWithCuda(sJ *output, unsigned int size, TYPE digit)
	{
		sJ *dev_c = 0;
		sJ* c = new sJ[1];
		sJ *dev_ex = 0;

		hipError_t cudaStatus;

		uint64 launchWidth, neededLaunches, currentLaunch;

		uint64 lastWrite = 0;

		// Choose which GPU to run on, change this on a multi-GPU system.
		cudaStatus = hipSetDevice(gpu);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		// Allocate GPU buffer for temp vector
		cudaStatus = hipMalloc((void**)&dev_ex, this->size * sizeof(sJ) * 7);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		// Allocate GPU buffer for output vector
		cudaStatus = hipMalloc((void**)&dev_c, this->size * sizeof(sJ) * 7);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		launchWidth = (uint64)this->size * strideMultiplier;

		//need to round up
		//because bbp condition for stopping is <= digit, number of total elements in summation is 1 + digit
		//even when digit/launchWidth is an integer, it is necessary to add 1
		neededLaunches = ((this->data->sumEnd - this->data->beginFrom) / launchWidth) + 1LLU;
		while ( (currentLaunch = this->prog->launchCount++) < neededLaunches) {

			uint64 begin = this->data->beginFrom + (launchWidth * currentLaunch);
			uint64 end = this->data->beginFrom + (launchWidth * (currentLaunch + 1)) - 1;
			if (end > this->data->sumEnd) end = this->data->sumEnd;

			//after exactly cachePeriod number of launches since last period between all gpus, write all data computed during and before the period to status buffer for progress thread to save
			if ((currentLaunch - lastWrite) >= cachePeriod) {

				lastWrite += cachePeriod;

				//copy current results into temp array to reduce and update status
				cudaStatus = hipMemcpy(dev_ex, dev_c, size * sizeof(sJ) * 7, hipMemcpyDeviceToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed in status update!\n");
					goto Error;
				}

				cudaStatus = reduceSJ(dev_ex, size * 7);

				if (cudaStatus != hipSuccess) {
					goto Error;
				}

				// Copy result (reduced into first element) from GPU buffer to host memory.
				cudaStatus = hipMemcpy(c, dev_ex, 1 * sizeof(sJ), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMemcpy failed in status update!\n");
					goto Error;
				}

				this->prog->queueMtx[this->gpu].lock();
				this->prog->currentResult[this->gpu].emplace_back(c[0], this->data->beginFrom + (launchWidth * lastWrite));
				this->prog->queueMtx[this->gpu].unlock();
			}

			// Launch a kernel on the GPU with one thread for each element.
			bbpKernel << <blockCount * 7, threadCountPerBlock >> > (dev_c, this->prog->deviceProg, this->data->startingExponent, begin, end, strideMultiplier);

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "bbpKernel launch failed on gpu%d: %s\n", this->gpu, hipGetErrorString(cudaStatus));
				goto Error;
			}

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bbpKernel on gpu %d!\n", cudaStatus, this->gpu);
				goto Error;
			}

			//give the rest of the computer some gpu time to reduce system choppiness
			if (primaryGpu) {
				std::this_thread::sleep_for(std::chrono::milliseconds(1));
			}
		}

		cudaStatus = reduceSJ(dev_c, size * 7);

		if (cudaStatus != hipSuccess) {
			goto Error;
		}

		// Copy result (reduced into first element) from GPU buffer to host memory.
		cudaStatus = hipMemcpy(c, dev_c, 1 * sizeof(sJ), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!\n");
			goto Error;
		}

		this->output = c[0];

	Error:
		free(c);
		hipFree(dev_c);
		hipFree(dev_ex);

		this->error = cudaStatus;
	}
};

int bbpLauncher::totalLaunchers = 0;

//uses 32 bit multiplications to compute the highest 64 and lowest 64 bits of squaring a 64 bit number
//in assembly in order to access carry bit
//saves work with realization that (hi + lo)^2 = hi^2 + 2*hi*lo + lo^2
//compare to non-squaring multiplication (hi1 + lo1)*(hi2 + lo2) = hi1*hi2 + hi1*lo2 + lo1*hi2 + lo1*lo2
//one fewer multiplication is needed
__device__ void square64By64(uint64 multiplicand, uint64 * lo, uint64 * hi) {

	asm("{\n\t"
		".reg .u64          m0, m1, m2;\n\t"
		".reg .u32          t0, t1, t2, t3, v0, v1;\n\t"
		"mov.b64           {v0, v1}, %2;\n\t" //splits a into hi and lo 32 bit words
		"mul.wide.u32       m0, v0, v0;    \n\t" //m0 = alo*alo
		"mul.wide.u32       m1, v0, v1;    \n\t" //m1 = alo*ahi
		"mul.wide.u32       m2, v1, v1;    \n\t" //m2 = ahi*ahi
		"mov.b64           {t0, t1}, m0;\n\t"
		"mov.b64           {t2, t3}, m2;\n\t"
		"add.cc.u64         m1, m1, m1;\n\t" //because (ahi + alo)^2 = ahi^2 + 2*alo*ahi + alo^2, we must double m1
		"addc.u32           t3,  t3,  0;\n\t"
		"mov.b64           {v0, v1}, m1;\n\t"
		"add.cc.u32         t1, t1, v0;\n\t"
		"addc.cc.u32        t2, t2, v1;\n\t"
		"addc.u32           t3, t3, 0;\n\t"
		"mov.b64            %0, {t0, t1};\n\t" //concatenates t0 and t1 into 1 64 bit word
		"mov.b64            %1, {t2, t3};\n\t" //concatenates t2 and t3 into 1 64 bit word
		"}"
		: "=l"(*lo), "=l"(*hi)
		: "l"(multiplicand));
}

__device__ void subtractModIfMoreThanMod(uint64 & value, const uint64 & mod) {
	asm("{\n\t"
		".reg .u64        t0;\n\t"
		"sub.u64          t0, %1, %2;\n\t"
		"min.u64          %0, t0, %1;\n\t"
		"}"
		: "=l"(value)
		: "l"(value), "l"(mod));
}

//using R=2^32, performs a 2 step montgomery reduction on the 128-bit number represented by hi and lo
//assembly is used to access carry bit
__device__ void montgomeryAddAndShift32Bit(uint64 & hi, uint64 & lo, const uint64 & mod, const uint32 & mprime) {
	//a : multiplicand
	//b : multiplier
	//_lo : low 32 bits of result
	//_hi : high 32 bits of result
	asm("{\n\t"
		".reg .u32          t0, t1, t2, t3, z0, m0, m1;\n\t"
		"mov.b64           {m0, m1}, %3;\n\t" //splits mod into m0 and m1
		"mov.b64           {t0, t1}, %1;\n\t" //splits lo into hi and lo 32 bit words
		"mov.b64           {t2, t3}, %2;\n\t" //splits hi into hi and lo 32 bit words

		//montgomery reduction on least significant 32-bit word
		"mul.lo.u32         z0, %4, t0;\n\t"
		"mad.lo.cc.u32      t0, z0, m0, t0;\n\t"
		"madc.hi.cc.u32     t1, z0, m0, t1;\n\t"
		"addc.cc.u32        t2,  0, t2;\n\t"
		"addc.u32           t3,  0, t3;\n\t"
		"mad.lo.cc.u32      t1, z0, m1, t1;\n\t"
		"madc.hi.cc.u32     t2, z0, m1, t2;\n\t"
		"addc.u32           t3,  0, t3;\n\t"

		//montgomery reduction on second least significant 32-bit word
		"mul.lo.u32         z0, %4, t1;\n\t"
		"mad.lo.cc.u32      t1, z0, m0, t1;\n\t"
		"madc.hi.cc.u32     t2, z0, m0, t2;\n\t"
		"addc.u32           t3,  0, t3;\n\t"
		"mad.lo.cc.u32      t2, z0, m1, t2;\n\t"
		"madc.hi.u32        t3, z0, m1, t3;\n\t"
		"mov.b64            %0, {t2, t3};\n\t" //concatenates t2 and t3 into 1 64 bit word
		"}"
		: "=l"(hi)
		: "l"(lo), "l"(hi), "l"(mod), "r"(mprime));
}

__device__ void add128Bit(uint64 & addendHi, uint64 & addendLo, uint64 augendHi, uint64 augendLo) {
	asm("{\n\t"
		"add.cc.u64         %1, %3, %5;\n\t"
		"addc.u64           %0, %2, %4;\n\t"
		"}"
		: "=l"(addendHi), "=l"(addendLo)
		: "l"(addendHi), "l"(addendLo), "l"(augendHi), "l"(augendLo));
}

__device__ void sub128Bit(uint64 & addendHi, uint64 & addendLo, uint64 augendHi, uint64 augendLo) {
	asm("{\n\t"
		"sub.cc.u64         %1, %3, %5;\n\t"
		"subc.u64           %0, %2, %4;\n\t"
		"}"
		: "=l"(addendHi), "=l"(addendLo)
		: "l"(addendHi), "l"(addendLo), "l"(augendHi), "l"(augendLo));
}

//finds output such that (n * output) % 2^64 = -1
//found this approach used here: http://plouffe.fr/simon/1-s2.0-S0167819118300334-main.pdf
//an explanation of the approach: http://marc-b-reynolds.github.io/math/2017/09/18/ModInverse.html
//saves from 15-25% of the total computation time over xbinGCD method (on the lower side of that for larger digit computations)
__device__ void modInverseNewtonsMethod(uint64 n, uint64 & output) {
	//n * 3 xor 2
	output = ((n << 1) + n) ^ 2LLU;

#pragma unroll
	for (int i = 0; i < 4; i++) {
		output = output * (2 - (n * output));
	}

	//we have (n * output) % 2^64 = 1, so we need to invert it
	output = -output;
}

//montgomery multiplication routine identical to above except for only being used when abar and bbar are known in advance to be the same
//uses a faster multiplication routine for squaring than is possible while not squaring
__device__ void montgomerySquare(uint64 abar, uint64 mod, uint32 mprime, uint64 & output) {

	uint64 tlo = 0;// , tm = 0;

	square64By64(abar, &tlo, &output);

	montgomeryAddAndShift32Bit(output, tlo, mod, mprime);

	//can be removed if mod < 2^62
	//see this paper: https://pdfs.semanticscholar.org/0e6a/3e8f30b63b556679f5dff2cbfdfe9523f4fa.pdf
#ifdef QUINTILLION
	subtractModIfMoreThanMod(output, mod);
#endif
}

__device__ void fixedPointDivisionExact(const uint64 & mod, const uint64 & r, const uint64 & mPrime, uint64 * result, int negative) {
	if (!r) return;

	uint64 q0 = (-r)*mPrime;
	uint64 q1 = -(1LLU) - __umul64hi(mod, q0);
	q1 *= mPrime;

	if(!negative) add128Bit(result[1], result[0], q1, q0);
	else sub128Bit(result[1], result[0], q1, q0);
}

__device__ void fixedPointDivisionExactWithShift(const uint64 & mod, const uint64 & r, const uint64 & mPrime, uint64 * result, int shift, int negative) {
	if (!r) return;

	uint64 q0 = (-r)*mPrime;
	uint64 q1 = -(1LLU) - __umul64hi(mod, q0);
	q1 *= mPrime;

	q0 >>= shift;
	if(shift <= 64) q0 = q0 + (q1 << (64 - shift));
	else q0 = q0 + (q1 >> (shift - 64));
	q1 >>= shift;

	if (!negative) {
		result[0] += q0;
		result[1] += q1;
		if (result[0] < q0) result[1]++;
	}
	else {
		uint64 check = result[0];
		result[0] -= q0;
		result[1] -= q1;
		if (result[0] > check) result[1]--;
	}

}

//using left-to-right binary exponentiation
//the position of the highest bit in exponent is passed into the function as a parameter (it is more efficient to find it outside)
//uses montgomery multiplication to reduce difficulty of modular multiplication (runs in 55% of runtime of non-montgomery modular multiplication)
//montgomery multiplication suggested by njuffa
//adds the 128 bit number representing ((2^exp)%mod)/mod to result
__device__ __noinline__ void modExpLeftToRight(uint64 exp, const uint64 & mod, uint64 * result, const int & negative, uint64 montgomeryStart) {
	uint64 output = 1;
	uint64 mPrime;

	modInverseNewtonsMethod(mod, mPrime);

	uint32 mPrime32 = mPrime;

	//exp = exp - subtract;

	int shift = 0;

	if (exp < 128) {
		shift = 128 - exp;
		exp = 128;
	}

	//this makes it unnecessary to convert out of montgomery space
	exp -= 64;

	output = montgomeryStart;

	int shiftToLittleBit = 63 - __clzll(exp);

	while (shiftToLittleBit-- != 0) {

		montgomerySquare(output, mod, mPrime32, output);
		
		output <<= (exp >> shiftToLittleBit) & 1;

	}

	//remove these if you don't mind a slight decrease in precision
#ifndef QUINTILLION
	subtractModIfMoreThanMod(output, mod << 1);
#endif
	subtractModIfMoreThanMod(output, mod);

	if (shift) {
		fixedPointDivisionExactWithShift(mod, output, -mPrime, result, shift, negative);
	}
	else {
		fixedPointDivisionExact(mod, output, -mPrime, result, negative);
	}
}

//finds montgomeryStart so that 2^65 % startMod = montgomeryStart
//finds div so that montgomeryStart + n*div is congruent to 2^65 % (startMod - n*modCoefficient)
//this is possible because montgomery multiplication does not require we know 2^65 % mod exactly, but requires we know a number congruent to 2^65 % mod (as long as this number is less than 2^63)
//div is inversely proportional to startMod ( div = 2^65 / startMod )
//montgomeryStart + n*div is < 2*mod for mod > 2^(32.5 + log(n))
__device__ __noinline__ void fastModApproximator(uint64 startMod, uint64 modCoefficient, uint64 & montgomeryStart, uint64 & div) {
		div = twoTo63Power / startMod;
		div <<= 1;
		if (-(div * startMod) > startMod) div++;
		div <<= 1;
		if (-(div * startMod) > startMod) div++;
		montgomeryStart = 0 - (div * startMod);// 2^65 - div*startMod = 2^65 % startMod
		div *= modCoefficient;
}

//computes strideMultiplier # of summation terms
__device__ void bbp(uint64 startingExponent, uint64 start, uint64 end, uint64 strideMultiplier, uint64 startingMod, uint64 modCoefficient, int negative, sJ* output, uint64* progress, int progressCheck) {

	//find 2 in montgomery space
	uint64 startMod = modCoefficient * end + startingMod;
	uint64 montgomeryStart, div;

	fastModApproximator(startMod, modCoefficient, montgomeryStart, div);
	
	//go backwards so we can add div instead of subtracting it
	//subtracting produces a likelihood of underflow (whereas addition will not cause overflow for any mod where 2^8 < mod < (2^64 - 2^8) )
	for (uint64 k = end; k >= start && k <= end; k--) {
		uint64 exp = startingExponent - (k*10LLU);
		uint64 mod = modCoefficient * k + startingMod;
		if(startMod <= fastModLimit) {
			montgomeryStart = twoTo63Power % mod;
			montgomeryStart <<= 1;
			subtractModIfMoreThanMod(montgomeryStart, mod);
			montgomeryStart <<= 1;
			subtractModIfMoreThanMod(montgomeryStart, mod);
		}

		modExpLeftToRight(exp, mod, output->s, negative, montgomeryStart);

		negative ^= 1;
		montgomeryStart += div;
	}

	if ((start & 0xffff) == 0) {
		//printf("%llu\n", exp);
		//only 1 thread (with gridId 0 on GPU0) ever updates the progress
		//*progress = end;
		atomicMax(progress, end);
	}
}

//determine from thread and block position which parts of summation to calculate
//only one of the threads per kernel (AND ONLY ON GPU0) will report progress
//stride over all parts of summation in bbp formula where k <= startingExponent (between all threads of all launches)
__global__ void bbpKernel(sJ *c, uint64 *progress, uint64 startingExponent, uint64 begin, uint64 end, uint64 strideMultiplier)
{
	int gridId = threadIdx.x + blockDim.x * blockIdx.x;
	uint64 start = begin + (gridId / 7)*strideMultiplier;
	uint64 mod = 0, modCoefficient = 4;
	end = ullmin(end, start + strideMultiplier - 1);
	int negative = end & 1;
	switch (gridId % 7) {
	case 0:
		mod = 1;//4k + 1
		startingExponent -= 3;
		negative ^= 1;
		break;
	case 1:
		mod = 3;//4k + 3
		startingExponent -= 8;
		negative ^= 1;
		break;
	case 2:
		mod = 1;//10k + 1
		modCoefficient = 10;
		break;
	case 3:
		mod = 3;//10k + 3
		modCoefficient = 10;
		startingExponent -= 2;
		negative ^= 1;
		break;
	case 4:
		mod = 5;//10k + 5
		modCoefficient = 10;
		startingExponent -= 6;
		negative ^= 1;
		break;
	case 5:
		mod = 7;//10k + 7
		modCoefficient = 10;
		startingExponent -= 6;
		negative ^= 1;
		break;
	case 6:
		mod = 9;//10k + 9
		modCoefficient = 10;
		startingExponent -= 8;
	}
	bbp(startingExponent, start, end, strideMultiplier, mod, modCoefficient, negative, c + gridId, progress, !!gridId);
}

//stride over current leaves of reduce tree
__global__ void reduceSJKernel(sJ *c, int offset, int stop) {
	int stride = blockDim.x * gridDim.x;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	while (i < stop) {
		int augend = i + offset;
		sJAdd(c + i, c + augend);
		i += stride;
	}
}

//standard tree-based parallel reduce
hipError_t reduceSJ(sJ *c, unsigned int size) {
	hipError_t cudaStatus;
	while (size > 1) {
		int nextSize = (size + 1) >> 1;

		//size is odd
		if (size&1) reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize - 1);
		//size is even
		else reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceSJKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceSJKernel!\n", cudaStatus);
			return cudaStatus;
		}

		size = nextSize;
	}
	return cudaStatus;
}

int loadProperties() {
	std::cout << "Loading properties from " << propertiesFile << std::endl;
	FILE * propF = fopen(propertiesFile.c_str(), "r");

	if (propF == NULL) {
		std::cout << "Could not open " << propertiesFile << "!" << std::endl;
		return 1;
	}

	int readLines = 0;

	readLines += fscanf(propF, "%llu", &strideMultiplier);
	readLines += fscanf(propF, "%d", &blockCount);
	readLines += fscanf(propF, "%d", &primaryGpu);
	readLines += fscanf(propF, "%d", &benchmarkBlockCounts);
	readLines += fscanf(propF, "%d", &numRuns);
	readLines += fscanf(propF, "%llu", &benchmarkTarget);
	readLines += fscanf(propF, "%d", &startBlocks);
	readLines += fscanf(propF, "%d", &blocksIncrement);
	readLines += fscanf(propF, "%d", &incrementLimit);
	if (readLines != 9) {
		std::cout << "Properties loading failed!" << std::endl;
		return 1;
	}

	return 0;
}

int benchmark() {
	digitData data(benchmarkTarget);
	totalGpus = 1;
	progressData prog(totalGpus);
	if (prog.error != hipSuccess) return 1;
	bbpLauncher gpuData;
	gpuData.totalGpus = totalGpus;
	gpuData.initialize(&data, &prog);
	std::vector<std::pair<double, int>> timings;
	for (blockCount = startBlocks; blockCount <= (startBlocks + incrementLimit*blocksIncrement); blockCount += blocksIncrement) {
		double total = 0.0;
		for (int j = 0; j < numRuns; j++) {
			prog.launchCount = 0;
			chr::high_resolution_clock::time_point start = chr::high_resolution_clock::now();
			gpuData.size = threadCountPerBlock * blockCount;
			gpuData.launch();
			chr::high_resolution_clock::time_point end = chr::high_resolution_clock::now();
			total += chr::duration_cast<chr::duration<double>>(end - start).count();
		}
		double avg = total / (double)numRuns;
		std::cout << "Average for " << blockCount << " blocks is " << avg << " seconds." << std::endl;
		std::pair<double, int> timingPair(avg, blockCount);
		timings.push_back(timingPair);
	}
	std::sort(timings.begin(), timings.end());
	std::cout << "Fastest block counts:" << std::endl;
	for (int i = 0; i < 10; i++) {
		std::cout << timings.at(i).second << " blocks at " << timings.at(i).first << " seconds." << std::endl;
	}
	return 0;
}

int main() {

	if (loadProperties()) return 1;

	hipError_t cudaStatus = hipGetDeviceCount(&totalGpus);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount failed!\n");
		return 1;
	}
	if (!totalGpus) {
		fprintf(stderr, "No GPUs detected in system!\n");
		return 1;
	}

	if (benchmarkBlockCounts) {
		return benchmark();
	}

	const int arraySize = threadCountPerBlock * blockCount;
	uint64 hexDigitPosition;
	std::cout << "Input hexDigit to calculate (1-indexed):" << std::endl;
	std::cin >> hexDigitPosition;

	digitData data(hexDigitPosition);

	std::thread * handles = new std::thread[totalGpus];
	bbpLauncher * gpuData = new bbpLauncher[totalGpus];

	progressData prog(totalGpus);
	if (prog.error != hipSuccess) return 1;
	if (prog.checkForProgressCache(&data)) return 1;

	chr::high_resolution_clock::time_point start = chr::high_resolution_clock::now();
	prog.begin = &start;

	std::thread progThread(&progressData::progressCheck, &prog);

	for (int i = 0; i < totalGpus; i++) {
		gpuData[i].totalGpus = totalGpus;
		gpuData[i].size = arraySize;
		gpuData[i].initialize(&data, &prog);

		handles[i] = std::thread(&bbpLauncher::launch, gpuData + i);
	}

	sJ cudaResult = prog.previousCache;

	for (int i = 0; i < totalGpus; i++) {

		handles[i].join();

		cudaStatus = gpuData[i].error;
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaBbpLaunch failed on gpu%d!\n", i);
			return 1;
		}

		sJ output = gpuData[i].output;

		//sum results from gpus
		sJAdd(&cudaResult, &output);
	}

	//tell the progress thread to quit
	prog.quit = 1;

	progThread.join();

	delete[] handles;
	delete[] gpuData;

	//uint64 hexDigit = finalizeDigit(cudaResult, hexDigitPosition);

	chr::high_resolution_clock::time_point end = chr::high_resolution_clock::now();

	printf("pi at hexadecimal digit %llu is %016llX %016llX\n",
		hexDigitPosition, cudaResult.s[1], cudaResult.s[0]);

	//find time elapsed during runtime of program, and add it to recorded runtime of previous unfinished run
	printf("Computed in %.8f seconds\n", prog.previousTime + (chr::duration_cast<chr::duration<double>>(end - start)).count());

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}

	return 0;
}