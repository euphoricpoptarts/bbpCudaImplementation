#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>

struct sJ {
	double s1, s4, s5, s6;
};

hipError_t addWithCuda(sJ *c, unsigned int size, long long digit);

__device__ const long baseSystem = 16;

__device__ int printOnce = 0;

//binary search to find highest 1 bit in multiplier
__device__ void findMultiplierHighestBit(const unsigned long long multiplier, unsigned long long *output) {
	
	//if no bits are 1 then highest bit doesn't exist
	if (!multiplier) {
		*output = 0;
		return;
	}

	int highestBitLocMax = 63;
	int highestBitLocMin = 0;

	int middle = (highestBitLocMax + highestBitLocMin) >> 1;

	unsigned long long highestBit = 1L;
	highestBit <<= middle;

	int less = highestBit <= multiplier;

	while (!((highestBit << 1) > multiplier && less)) {
		if (less) highestBitLocMin = middle + 1;
		else highestBitLocMax = middle - 1;
		middle = (highestBitLocMax + highestBitLocMin) >> 1;
		//this might not look necessary but it is
		highestBit = 1L;
		highestBit <<= middle;
		less = highestBit <= multiplier;
	}

	/*unsigned long long highestBit2 = 0x8000000000000000;

	while (highestBit2 > multiplier) highestBit2 >>= 1;

	if (highestBit != highestBit2 && !printOnce) {
		printf("multiplier %d error; highestBit %d; highestBit2 %d\n", multiplier, highestBit, highestBit2);
		printOnce = 1;
	}*/

	*output = highestBit;
}

__device__ void modMultiplyLeftToRight(const unsigned long long multiplicand, const unsigned long long multiplier, unsigned long long mod, unsigned long long *output) {
	unsigned long long result = multiplicand;

	//only perform modulus operation during loop if result is >= 2^61 (in order to prevent overflowing)
	const unsigned long long modCond = 0x2000000000000000;//2^61

	unsigned long long highestBitMask = 0;

	findMultiplierHighestBit(multiplier, &highestBitMask);

	while (highestBitMask > 1) {
		if (result >= modCond) result %= mod;
		result <<= 1;
		highestBitMask >>= 1;
		if (multiplier&highestBitMask)	result += multiplicand;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//perform right-to-left binary exponention taking modulus of both base and result at each step
//64 bit integers are required to accurately find the modular exponents of numbers when mod is >= ~10e6
//however, with CUDA 64 bit integers are implemented at compile time as two 32 bit integers
//this produces about a 10x slowdown over computations using 32 bit integers
__device__ void modExp(unsigned long long base, long exp, long mod, long *output) {
	const unsigned long mask = 1;
	unsigned long long result = 1;

	//only perform modulus operation during loop if result or base is >= 2^32 (in order to prevent either from overflowing)
	//this saves 30% computation time over performing modulus in every loop iteration
	const unsigned long long modCond = 0x100000000;//2^32

	while (exp > 0) {
		if (exp&mask) {
			result *= base;
			if (result >= modCond) result %= mod;
		}
		base *= base;
		if (base >= modCond) base %= mod;
		exp >>= 1;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//using left-to-right binary exponentiation
//the position of the highest bit in exponent is passed into the function as a parameter (it is more efficient to find it outside)
//this version allows base to be constant, thus reducing total number of moduli which must be calculated
//geometric mean of multiplication inputs is also substantially lower, allowing faster average multiplications
__device__ void modExpLeftToRight(const unsigned long long base, const unsigned long long exp, unsigned long long mod, unsigned long long highestBitMask, long long *output) {
	unsigned long long result = base;

	////only perform modulus operation during loop if result is >= 2^29 (in order to prevent overflowing)
	//const unsigned long long modCond = 0x20000000;//2^29

	while (highestBitMask > 1) {
		//if (result >= modCond) result %= mod;
		modMultiplyLeftToRight(result, result, mod, &result);//result *= result;
		highestBitMask >>= 1;
		if (exp&highestBitMask)	modMultiplyLeftToRight(result, base, mod, &result);//result *= base;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	//result %= mod;
	*output = result;
}

//find ( 16^n % mod ) / mod and add to partialSum
__device__ void fractionalPartOfSum(long long exp, long long mod, double *partialSum, long long highestBitMask) {
	long long expModResult = 0;
	modExpLeftToRight(baseSystem, exp, mod, highestBitMask, &expModResult);
	*partialSum += ((double)expModResult) / ((double)mod);
}

//stride over all parts of summation in bbp formula where k <= n
//to compute partial sJ sums
__device__ void bbp(long long n, long long start, long long stride, sJ *output) {

	double s1 = 0.0, s4 = 0.0, s5 = 0.0, s6 = 0.0;
	double trash = 0.0;
	long long highestExpBit = 1;
	while (highestExpBit <= n)	highestExpBit <<= 1;
	for (long long k = start; k <= n; k += stride) {
		while (highestExpBit > (n - k))  highestExpBit >>= 1;
		long long mod = 8 * k + 1;
		fractionalPartOfSum(n - k, mod, &s1, highestExpBit);
		mod += 3;
		fractionalPartOfSum(n - k, mod, &s4, highestExpBit);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s5, highestExpBit);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s6, highestExpBit);
		//remove any integer part of s1-s6
		s1 = modf(s1, &trash);
		s4 = modf(s4, &trash);
		s5 = modf(s5, &trash);
		s6 = modf(s6, &trash);
	}
	output[start].s1 = s1;
	output[start].s4 = s4;
	output[start].s5 = s5;
	output[start].s6 = s6;
}

//determine from thread and block position where to begin stride
//and how wide stride is
__global__ void bbpKernel(sJ *c, long digit)
{
	long long stride = blockDim.x * gridDim.x;
	long long i = threadIdx.x + blockDim.x * blockIdx.x;
	bbp(digit, i, stride, c);
}

//stride over current leaves of reduce tree
__global__ void reduceSJKernel(sJ *c, int offset, int stop) {
	int stride = blockDim.x * gridDim.x;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	while (i < stop) {
		int augend = i + offset;
		c[i].s1 += c[augend].s1;
		c[i].s4 += c[augend].s4;
		c[i].s5 += c[augend].s5;
		c[i].s6 += c[augend].s6;
		i += stride;
	}
}

//standard tree-based parallel reduce
hipError_t reduceSJ(sJ *c, unsigned int size) {
	hipError_t cudaStatus;
	while (size > 1) {
		int nextSize = (size + 1) >> 1;

		//size is odd
		if (size&1) reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize - 1);
		//size is even
		else reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceSJKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceSJKernel!\n", cudaStatus);
			return cudaStatus;
		}

		size = nextSize;
	}
	return cudaStatus;
}

//compute four steps of sJ sums for i > n and add to sJ sums found previously
//combine sJs according to bbp formula
//multiply by 16^5 to extract five digits of pi starting at n
long finalizeDigit(sJ input, long long n) {
	double reducer = 1.0;
	double s1 = input.s1;
	double s4 = input.s4;
	double s5 = input.s5;
	double s6 = input.s6;
	double trash = 0.0;
	if (n < 16000) {
		for (int i = 0; i < 4; i++) {
			n++;
			reducer /= (double)baseSystem;
			double eightN = 8.0 * n;
			s1 += reducer / (eightN + 1.0);
			s4 += reducer / (eightN + 4.0);
			s5 += reducer / (eightN + 5.0);
			s6 += reducer / (eightN + 6.0);
		}
	}
	//remove any integer part of s1-s6
	s1 = modf(s1, &trash);
	s4 = modf(s4, &trash);
	s5 = modf(s5, &trash);
	s6 = modf(s6, &trash);
	double hexDigit = 4.0*s1 - 2.0*s4 - s5 - s6;
	hexDigit = modf(hexDigit, &trash);
	if (hexDigit < 0) hexDigit++;
	hexDigit *= baseSystem*baseSystem*baseSystem*baseSystem*baseSystem;
	printf("hexDigit = %.8f\n", hexDigit);
	return (long)hexDigit;
}

int main()
{
	const int arraySize = 128 * 128;
	const long long digitPosition = 999999999;
	sJ c[arraySize];

	clock_t start = clock();

	hipError_t cudaStatus = addWithCuda(c, arraySize, digitPosition);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	long hexDigit = finalizeDigit(c[0], digitPosition);

	clock_t end = clock();

	printf("pi at hexadecimal digit %d is %X\n",
		digitPosition + 1, hexDigit);

	printf("Computed in %.8f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA
hipError_t addWithCuda(sJ *c, unsigned int size, long long digit)
{
	sJ *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffer for output vector    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(sJ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	bbpKernel << <128, 128 >> >(dev_c, digit);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bbpKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bbpKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	sJ expected;

	expected.s1 = 0;
	expected.s4 = 0;
	expected.s5 = 0;
	expected.s6 = 0;

	for (int j = 0; j < size; j++) {
		expected.s1 += c[j].s1;
		expected.s4 += c[j].s4;
		expected.s5 += c[j].s5;
		expected.s6 += c[j].s6;
	}

	cudaStatus = reduceSJ(dev_c, size);

	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	if (fabs(c[0].s1 - expected.s1) > 1e-10) {
		printf("s1 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s1, c[0].s1);
	}
	if (fabs(c[0].s4 - expected.s4) > 1e-10) {
		printf("s4 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s4, c[0].s4);
	}
	if (fabs(c[0].s5 - expected.s5) > 1e-10) {
		printf("s5 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s5, c[0].s5);
	}
	if (fabs(c[0].s6 - expected.s6) > 1e-10) {
		printf("s6 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s6, c[0].s6);
	}

Error:
	hipFree(dev_c);

	return cudaStatus;
}
