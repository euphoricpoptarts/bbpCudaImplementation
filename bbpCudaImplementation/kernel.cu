#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <Windows.h>
#include <deque>

#define TYPE unsigned long long
#define INT_64 unsigned long long

struct sJ {
	double s1 = 0.0, s4 = 0.0, s5 = 0.0, s6 = 0.0;
};

typedef struct {
	volatile INT_64 *currentProgress;
	TYPE maxProgress;
	int quit = 0;
} PROGRESSDATA, *PPROGRESSDATA;

typedef struct {
	sJ output;
	INT_64 digit;
	int gpu = 0;
	int totalGpus = 0;
	int size = 0;
	hipError_t error;
} BBPLAUNCHERDATA, *PBBPLAUNCHERDATA;

DWORD WINAPI cudaBbpLauncher(LPVOID dataV);

//warpsize is 32 so optimal value is probably always a multiple of 32
const int threadCountPerBlock = 64;
//this is more difficult to optimize but seems to not like odd numbers
const int blockCount = 560;

__device__ const TYPE baseSystem = 16;
__device__ const int baseExpOf2 = 4;

__device__ const int typeSize = sizeof(TYPE) * 8 - 1;
__device__ const TYPE multiplyModCond = 0x4000000000000000;//2^62
//__device__ const int int64Size = sizeof(INT_64) * 8 - 1;
__device__ const INT_64 int64ModCond = 0x40000000;
__device__ const INT_64 int64MaxBit = 0x8000000000000000;

__device__ int printOnce = 0;

//not actually quick
__device__ void quickMod(INT_64 input, const INT_64 mod, INT_64 *output) {

	/*INT_64 copy = input;
	INT_64 test = input % mod;*/
	INT_64 temp = mod;
	while (temp < input && !(temp&int64MaxBit)) temp <<= 1;
	if (temp > input) temp >>= 1;
	while (input >= mod && temp >= mod) {
		if (input >= temp) input -= temp;
		temp >>= 1;
	}
	/*if (input != test && !atomicAdd(&printOnce,1))
	{
	printf("input %llu mod %llu error\n", copy, mod);
	printOnce = 1;
	}*/
	*output = input;
}

//binary search to find highest 1 bit in multiplier
__device__ void findMultiplierHighestBit(const TYPE multiplier, TYPE *output) {

	//if no bits are 1 then highest bit doesn't exist
	if (!multiplier) {
		*output = 0;
		return;
	}

	int highestBitLocMax = typeSize;
	int highestBitLocMin = 0;

	int middle = (highestBitLocMax + highestBitLocMin) >> 1;

	TYPE highestBit = 1L;
	highestBit <<= middle;

	int less = highestBit <= multiplier;

	while (!((highestBit << 1) > multiplier && less)) {
		if (less) highestBitLocMin = middle + 1;
		else highestBitLocMax = middle - 1;
		middle = (highestBitLocMax + highestBitLocMin) >> 1;
		//this might not look necessary but it is
		highestBit = 1L;
		highestBit <<= middle;
		less = highestBit <= multiplier;
	}

	/*unsigned long long highestBit2 = 0x8000000000000000;

	while (highestBit2 > multiplier) highestBit2 >>= 1;

	if (highestBit != highestBit2 && !printOnce) {
	printf("multiplier %d error; highestBit %d; highestBit2 %d\n", multiplier, highestBit, highestBit2);
	printOnce = 1;
	}*/

	*output = highestBit;
}

//hacker's delight method to find highest bit in a long long (it just works)
//http://graphics.stanford.edu/~seander/bithacks.html
//just barely faster than built-in CUDA __clzll
__device__ void findMultiplierHighestBitHackersDelight(TYPE multiplier, TYPE *output) {

	multiplier |= multiplier >> 1;
	multiplier |= multiplier >> 2;
	multiplier |= multiplier >> 4;
	multiplier |= multiplier >> 8;
	multiplier |= multiplier >> 16;
	multiplier |= multiplier >> 32;

	*output = multiplier ^ (multiplier >> 1);

}

__device__ void modMultiplyLeftToRight(const TYPE multiplicand, const TYPE multiplier, TYPE mod, TYPE *output) {
	TYPE result = multiplicand;

	TYPE highestBitMask = 0;

	findMultiplierHighestBitHackersDelight(multiplier, &highestBitMask);

	while (highestBitMask > 1) {
		//only perform modulus operation during loop if result is >= (TYPE maximum + 1)/4 (in order to prevent overflowing)
		if (result >= multiplyModCond) result %= mod;
		result <<= 1;
		highestBitMask >>= 1;
		if (multiplier&highestBitMask)	result += multiplicand;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

__device__ void modMultiplyRightToLeft(INT_64 multiplicand, INT_64 multiplier, INT_64 mod, INT_64 *output) {
	INT_64 result = 0;

	INT_64 mask = 1;

	while (multiplier > 0) {
		if (multiplier&mask) {
			result += multiplicand;

			//only perform modulus operation during loop if result is >= (INT_64 maximum + 1)/2 (in order to prevent overflowing)
			if (result >= int64MaxBit) result %= mod;
		}
		multiplicand <<= 1;
		if (multiplicand >= int64MaxBit) multiplicand %= mod;
		multiplier >>= 1;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//leverages a machine instruction that returns the highest 64 bits of the multiplication operation
//multiplicand and multiplier should always be less than mod (may work correctly even if this is not the case)
//maxMod is constant with respect to each mod, therefore best place to calculate is in modExp functions
__device__ void modMultiply64Bit(INT_64 multiplicand, INT_64 multiplier, INT_64 mod, INT_64 maxMod, INT_64 *output) {
	INT_64	hi = __umul64hi(multiplicand, multiplier);
	INT_64 result = (multiplicand * multiplier) % mod;
	while (hi) {
		INT_64 lo = hi * maxMod;

		//multiplyModCond should be (2^64)/(number of loop iterations)
		//where loop iterations are roughly 64/(64 - log2(mod))
		//THEREFORE THIS SHOULD NOT BE A COMPILE TIME CONSTANT
		//but a runtime variable set at launch based upon the maximum mod that will be passed to this function
		//for 2^40 number of loops is 2, for 2^50 number of loops is 4
		if (lo > multiplyModCond) lo %= mod;

		result +=  lo;
		hi = __umul64hi(hi, maxMod);
	}
	*output = result % mod;
}

//leverages a machine instruction that returns the highest 64 bits of the multiplication operation
//multiplicand and multiplier should always be less than mod (may work correctly even if this is not the case)
//uses bitshifts to avoid multiplications inside the loop
//slower than other version (but could be faster if mod is close to 2^63)
__device__ void modMultiply64BitAlt(INT_64 multiplicand, INT_64 multiplier, INT_64 mod, INT_64 *output) {
	INT_64	hi = __umul64hi(multiplicand, multiplier);
	INT_64 result = (multiplicand * multiplier) % mod;
	while (hi) {

		//determine the number of bits to shift so that hi >= 2^63
		int leading = __clzll(hi);

		//hi is the highest 64 bits of multiplicand*multiplier
		//so hi is actually hi*2^64
		//shifting by leading does not change the true value, as (hi*2^leading)*2^(64 - leading) = hi*2^64
		hi <<= leading;

		hi %= mod;

		INT_64 lo = hi << (64 - leading);
		hi >>= leading;

		//multiplyModCond should be (2^64)/(number of loop iterations)
		//where loop iterations are roughly 64/(64 - log2(mod))
		//THEREFORE THIS SHOULD NOT BE A COMPILE TIME CONSTANT
		//but a runtime variable set at launch based upon the maximum mod that will be passed to this function
		//for 2^40 number of loops is 2, for 2^50 number of loops is 4
		if (lo > multiplyModCond) lo %= mod;

		result += lo;
	}
	*output = result % mod;
}

//perform right-to-left binary exponention taking modulus of both base and result at each step
//64 bit integers are required to accurately find the modular exponents of numbers when mod is >= ~10e6
//however, with CUDA 64 bit integers are implemented at compile time as two 32 bit integers
//this produces about a 10x slowdown over computations using 32 bit integers
__device__ void modExp(unsigned long long base, long exp, long mod, long *output) {
	const unsigned long mask = 1;
	unsigned long long result = 1;

	//only perform modulus operation during loop if result or base is >= 2^32 (in order to prevent either from overflowing)
	//this saves 30% computation time over performing modulus in every loop iteration
	const unsigned long long modCond = 0x100000000;//2^32

	while (exp > 0) {
		if (exp&mask) {
			result *= base;
			if (result >= modCond) result %= mod;
		}
		base *= base;
		if (base >= modCond) base %= mod;
		exp >>= 1;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//using left-to-right binary exponentiation
//the position of the highest bit in exponent is passed into the function as a parameter (it is more efficient to find it outside)
//this version allows base to be constant, thus reducing total number of moduli which must be calculated
//geometric mean of multiplication inputs is also substantially lower, allowing faster average multiplications
__device__ void modExpLeftToRight(const TYPE exp, TYPE mod, TYPE highestBitMask, TYPE *output) {
	INT_64 result = baseSystem;

	//only perform modulus operation during loop if result is >= sqrt((BIG_TYPE maximum + 1)/8) (in order to prevent overflowing)
	//INT_64 modCond = int64ModCond;

	INT_64 maxMod = int64MaxBit % mod;

	maxMod <<= 1;

	if (maxMod > mod) maxMod %= mod;

	while (highestBitMask > 1) {
		//this is not necessary as modMultiplyLeftToRight ensures result never overflows a 64 bit buffer
		//however performing this modulus saves time (more is less)
		//likely saves performing some moduli in modMultiplyLeftToRight or reduces overall size of arguments
		if (result >= mod) result %= mod;//quickMod(result, mod, &result);

		modMultiply64Bit(result, result, mod, maxMod, &result);//result *= result;
		highestBitMask >>= 1;
		if (exp&highestBitMask)	result <<= baseExpOf2;//modMultiplyLeftToRight(result, base, mod, &result);//result *= base;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	//result %= mod;//quickMod(result, mod, &result);
	*output = result;
}

//find ( 16^n % mod ) / mod and add to partialSum
__device__ void fractionalPartOfSum(TYPE exp, TYPE mod, double *partialSum, TYPE highestBitMask) {
	TYPE expModResult = 0;
	modExpLeftToRight(exp, mod, highestBitMask, &expModResult);
	*partialSum += ((double)expModResult) / ((double)mod);
}

//stride over all parts of summation in bbp formula where k <= n
//to compute partial sJ sums
__device__ void bbp(TYPE n, TYPE start, INT_64 end, int gridId, TYPE stride, sJ *output, volatile INT_64 *progress, int progressCheck) {

	double s1 = output[gridId].s1, s4 = output[gridId].s4, s5 = output[gridId].s5, s6 = output[gridId].s6;
	double trash = 0.0;
	TYPE highestExpBit = 1;
	while (highestExpBit <= n)	highestExpBit <<= 1;
	for (TYPE k = start; k <= end; k += stride) {
		while (highestExpBit > (n - k))  highestExpBit >>= 1;
		TYPE mod = 8 * k + 1;
		fractionalPartOfSum(n - k, mod, &s1, highestExpBit);
		mod += 3;
		fractionalPartOfSum(n - k, mod, &s4, highestExpBit);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s5, highestExpBit);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s6, highestExpBit);
		//remove any integer part of s1-s6
		s1 = modf(s1, &trash);
		s4 = modf(s4, &trash);
		s5 = modf(s5, &trash);
		s6 = modf(s6, &trash);
		if (!progressCheck) {
			//only 1 thread ever updates the progress
			*progress = k;
		}
	}
	output[gridId].s1 = s1;
	output[gridId].s4 = s4;
	output[gridId].s5 = s5;
	output[gridId].s6 = s6;
}

//determine from thread and block position where to begin stride
//only one of the threads per kernel (AND ONLY ON GPU0) will report progress
__global__ void bbpKernel(sJ *c, volatile INT_64 *progress, TYPE digit, int gpuNum, INT_64 begin, INT_64 end, INT_64 stride)
{
	int gridId = threadIdx.x + blockDim.x * blockIdx.x;
	TYPE start = begin + gridId + blockDim.x * gridDim.x * gpuNum;
	int progressCheck = gridId + blockDim.x * gridDim.x * gpuNum;
	bbp(digit, start, end, gridId, stride, c, progress, progressCheck);
}

//stride over current leaves of reduce tree
__global__ void reduceSJKernel(sJ *c, int offset, int stop) {
	int stride = blockDim.x * gridDim.x;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	while (i < stop) {
		int augend = i + offset;
		c[i].s1 += c[augend].s1;
		c[i].s4 += c[augend].s4;
		c[i].s5 += c[augend].s5;
		c[i].s6 += c[augend].s6;
		i += stride;
	}
}

//standard tree-based parallel reduce
hipError_t reduceSJ(sJ *c, unsigned int size) {
	hipError_t cudaStatus;
	while (size > 1) {
		int nextSize = (size + 1) >> 1;

		//size is odd
		if (size&1) reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize - 1);
		//size is even
		else reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceSJKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceSJKernel!\n", cudaStatus);
			return cudaStatus;
		}

		size = nextSize;
	}
	return cudaStatus;
}

//compute four steps of sJ sums for i > n and add to sJ sums found previously
//combine sJs according to bbp formula
//multiply by 16^5 to extract five digits of pi starting at n
long finalizeDigit(sJ input, TYPE n) {
	double reducer = 1.0;
	double s1 = input.s1;
	double s4 = input.s4;
	double s5 = input.s5;
	double s6 = input.s6;
	double trash = 0.0;
	if (n < 16000) {
		for (int i = 0; i < 4; i++) {
			n++;
			reducer /= (double)baseSystem;
			double eightN = 8.0 * n;
			s1 += reducer / (eightN + 1.0);
			s4 += reducer / (eightN + 4.0);
			s5 += reducer / (eightN + 5.0);
			s6 += reducer / (eightN + 6.0);
		}
	}
	//remove any integer part of s1-s6
	s1 = modf(s1, &trash);
	s4 = modf(s4, &trash);
	s5 = modf(s5, &trash);
	s6 = modf(s6, &trash);
	double hexDigit = 4.0*s1 - 2.0*s4 - s5 - s6;
	hexDigit = modf(hexDigit, &trash);
	if (hexDigit < 0) hexDigit++;
	hexDigit *= baseSystem*baseSystem*baseSystem*baseSystem*baseSystem;
	printf("hexDigit = %.8f\n", hexDigit);
	return (long)hexDigit;
}

int main()
{
	try {
		const int arraySize = threadCountPerBlock * blockCount;
		const TYPE digitPosition = 9999999999;
		const int totalGpus = 2;
		HANDLE handles[totalGpus];
		BBPLAUNCHERDATA gpuData[totalGpus];

		clock_t start = clock();

		for (int i = 0; i < totalGpus; i++) {

			gpuData[i].digit = digitPosition;
			gpuData[i].gpu = i;
			gpuData[i].totalGpus = totalGpus;
			gpuData[i].size = arraySize;


			handles[i] = CreateThread(NULL, 0, *cudaBbpLauncher, (LPVOID)&(gpuData[i]), 0, NULL);

			if (handles[i] == NULL) {
				fprintf(stderr, "gpu%dThread failed to launch\n", i);
				return 1;
			}
		}

		sJ cudaResult;

		hipError_t cudaStatus;

		for (int i = 0; i < totalGpus; i++) {

			WaitForSingleObject(handles[i], INFINITE);
			CloseHandle(handles[i]);

			hipError_t cudaStatus = gpuData[i].error;
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "cudaBbpLaunch failed on gpu%d!\n", i);
				return 1;
			}

			sJ output = gpuData[i].output;

			//sum results from gpus
			cudaResult.s1 += output.s1;
			cudaResult.s4 += output.s4;
			cudaResult.s5 += output.s5;
			cudaResult.s6 += output.s6;
		}

		long hexDigit = finalizeDigit(cudaResult, digitPosition);

		clock_t end = clock();

		printf("pi at hexadecimal digit %llu is %05X\n",
			digitPosition + 1, hexDigit);

		printf("Computed in %.8f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
	catch(...) {
		printf("oops xD\n");
		return 1;
	}
}

//this function is meant to be run by an independent thread to output progress to the console
DWORD WINAPI progressCheck(LPVOID data) {
	PPROGRESSDATA progP = (PPROGRESSDATA)data;

	double lastProgress = 0;

	std::deque<double> progressQ;

	while(!(*progP).quit) {
		double progress = (double)(*((*progP).currentProgress)) / (double)(*progP).maxProgress;

		progressQ.push_front(progress - lastProgress);

		if (progressQ.size() > 10) progressQ.pop_back();

		double progressAvg = 0.0;

		for (std::deque<double>::iterator it = progressQ.begin(); it != progressQ.end(); *it++) progressAvg += *it;

		progressAvg /= (double) progressQ.size();

		double timeEst = 0.1*(1.0 - progress) / (progressAvg);
		lastProgress = progress;
		printf("Current progress is %3.3f%%. Estimated total runtime remaining is %8.3f seconds.\n", 100.0*progress, timeEst);

		Sleep(100);
	}
	return 0;
}

// Helper function for using CUDA
DWORD WINAPI cudaBbpLauncher(LPVOID dataV)//hipError_t addWithCuda(sJ *output, unsigned int size, TYPE digit)
{
	PBBPLAUNCHERDATA data = (PBBPLAUNCHERDATA)dataV;
	int size = (*data).size;
	int gpu = (*data).gpu;
	int totalGpus = (*data).totalGpus;
	INT_64 digit = (*data).digit;
	sJ *dev_c = 0;
	sJ* c = new sJ[size];

	hipError_t cudaStatus;

	PROGRESSDATA threadData;
	HANDLE thread;
	//these variables are linked between host and device memory allowing each to communicate about progress
	volatile INT_64 *currProgHost, *currProgDevice;

	if (gpu == 0) {

		//allow device to map host memory for progress ticker
		cudaStatus = hipSetDeviceFlags(hipDeviceMapHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDeviceFlags failed with error: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// Allocate Host memory for progress ticker
		cudaStatus = hipHostAlloc((void**)&currProgHost, sizeof(INT_64), hipHostMallocMapped);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "cudaHostAalloc failed!");
			goto Error;
		}

		//create link between between host and device memory for progress ticker
		cudaStatus = hipHostGetDevicePointer((INT_64 **)&currProgDevice, (INT_64 *)currProgHost, 0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipHostGetDevicePointer failed!");
			goto Error;
		}

		*currProgHost = 0;

		threadData = { currProgHost, digit, 0 };

		thread = CreateThread(NULL, 0, *progressCheck, (LPVOID)&threadData, 0, NULL);

		if (thread == NULL) {
			fprintf(stderr, "progressCheck thread creation failed\n");
			goto Error;
		}
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(gpu);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffer for output vector    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(sJ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	INT_64 stride =  (INT_64) size * (INT_64) totalGpus;

	INT_64 launchWidth = stride * 128LLU;

	//need to round up
	//because bbp condition for stopping is <= digit, number of total elements in summation is 1 + digit
	//even when digit/launchWidth is an integer, it is necessary to add 1
	INT_64 neededLaunches = (digit / launchWidth) + 1LLU;

	for (INT_64 launch = 0; launch < neededLaunches; launch++) {

		INT_64 begin = launchWidth * launch;
		INT_64 end = launchWidth * (launch + 1) - 1;
		if (end > digit) end = digit;

		// Launch a kernel on the GPU with one thread for each element.
		bbpKernel << <blockCount, threadCountPerBlock >> > (dev_c, currProgDevice, digit, gpu, begin, end, stride);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "bbpKernel launch failed on gpu%d: %s\n", gpu, hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bbpKernel on gpu 1!\n", cudaStatus);
			goto Error;
		}

		//give the rest of the computer some gpu time to reduce system choppiness
		Sleep(1);
	}

	if (gpu == 0) {

		//tell the progress thread to quit
		threadData.quit = 1;

		WaitForSingleObject(thread, INFINITE);
		CloseHandle(thread);
	}

	cudaStatus = reduceSJ(dev_c, size);

	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	// Copy result vector from GPU 0 buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	(*data).output = c[0];

Error:
	free(c);
	hipFree(dev_c);

	(*data).error = cudaStatus;
	return 0;
}
