#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>

struct sJ {
	double s1, s4, s5, s6;
};

hipError_t addWithCuda(sJ *c, unsigned int size, long digit);

__device__ const long baseSystem = 16;

__device__ void modExp(long long base, long exp, long long mod, long *output) {
	long long mask = 1;
	long long result = 1;
	while (exp > 0) {
		if (exp&mask) result *= base;
		base *= base;
		result %= mod;
		base %= mod;
		exp >>= 1;
	}
	*output = result;
}

__device__ void fractionalPartOfSum(long exp, long mod, double *partialSum) {
	long expModResult = 0;
	modExp(baseSystem, exp, mod, &expModResult);
	*partialSum += ((double)expModResult) / ((double)mod);
}

__device__ void bbp(long n, long start, long stride, sJ *output) {

	double s1 = 0.0, s4 = 0.0, s5 = 0.0, s6 = 0.0;
	double trash = 0.0;
	for (long k = start; k <= n; k += stride) {
		long mod = 8 * k + 1;
		fractionalPartOfSum(n - k, mod, &s1);
		mod += 3;
		fractionalPartOfSum(n - k, mod, &s4);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s5);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s6);
		//remove any integer part of s1-s6
		s1 = modf(s1, &trash);
		s4 = modf(s4, &trash);
		s5 = modf(s5, &trash);
		s6 = modf(s6, &trash);
	}
	output[start].s1 = s1;
	output[start].s4 = s4;
	output[start].s5 = s5;
	output[start].s6 = s6;
}

__global__ void bbpKernel(sJ *c, long digit)
{
	int i = threadIdx.x;
	bbp(digit, i, blockDim.x, c);
}

__global__ void reduceSJKernel(sJ *c, int stride) {
	int i = threadIdx.x;
	int augend = i + stride;
	c[i].s1 += c[augend].s1;
	c[i].s4 += c[augend].s4;
	c[i].s5 += c[augend].s5;
	c[i].s6 += c[augend].s6;
}

//standard tree-based parallel reduce
hipError_t reduceSJ(sJ *c, unsigned int size) {
	hipError_t cudaStatus;
	while (size > 1) {
		int nextSize = (size + 1) / 2;

		//size is odd
		if (size&1) reduceSJKernel<< <1, nextSize - 1 >> >(c, nextSize);
		//size is even
		else reduceSJKernel<< <1, nextSize >> >(c, nextSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceSJKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceSJKernel!\n", cudaStatus);
			return cudaStatus;
		}

		size = nextSize;
	}
	return cudaStatus;
}

long finalizeDigit(sJ input, long n) {
	double reducer = 1.0;
	double s1 = input.s1;
	double s4 = input.s4;
	double s5 = input.s5;
	double s6 = input.s6;
	double trash = 0.0;
	if (n < 16000) {
		for (int i = 0; i < 4; i++) {
			n++;
			reducer /= (double)baseSystem;
			double eightN = 8.0 * n;
			s1 += reducer / (eightN + 1.0);
			s4 += reducer / (eightN + 4.0);
			s5 += reducer / (eightN + 5.0);
			s6 += reducer / (eightN + 6.0);
		}
	}
	//remove any integer part of s1-s6
	s1 = modf(s1, &trash);
	s4 = modf(s4, &trash);
	s5 = modf(s5, &trash);
	s6 = modf(s6, &trash);
	double hexDigit = 4.0*s1 - 2.0*s4 - s5 - s6;
	hexDigit = modf(hexDigit, &trash);
	if (hexDigit < 0) hexDigit++;
	hexDigit *= baseSystem*baseSystem*baseSystem*baseSystem*baseSystem;
	printf("hexDigit = %.8f\n", hexDigit);
	return (long)hexDigit;
}

int main()
{
	const int arraySize = 256;
	const long digitPosition = 1000000;
	sJ c[arraySize];

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, arraySize, digitPosition);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	long hexDigit = finalizeDigit(c[0], digitPosition);

	printf("pi at hexadecimal digit %d is %X\n",
		digitPosition, hexDigit);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA
hipError_t addWithCuda(sJ *c, unsigned int size, long digit)
{
	sJ *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffer for output vector    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(sJ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	bbpKernel << <1, size >> >(dev_c, digit);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bbpKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bbpKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	sJ expected;

	expected.s1 = 0;
	expected.s4 = 0;
	expected.s5 = 0;
	expected.s6 = 0;

	for (int j = 0; j < size; j++) {
		expected.s1 += c[j].s1;
		expected.s4 += c[j].s4;
		expected.s5 += c[j].s5;
		expected.s6 += c[j].s6;
	}

	cudaStatus = reduceSJ(dev_c, size);

	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	if (fabs(c[0].s1 - expected.s1) > 1e-10) {
		printf("s1 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s1, c[0].s1);
	}
	if (fabs(c[0].s4 - expected.s4) > 1e-10) {
		printf("s4 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s4, c[0].s4);
	}
	if (fabs(c[0].s5 - expected.s5) > 1e-10) {
		printf("s5 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s5, c[0].s5);
	}
	if (fabs(c[0].s6 - expected.s6) > 1e-10) {
		printf("s6 not correct\n");
		printf("Expected %.8f; Actual %.8f\n", expected.s6, c[0].s6);
	}

Error:
	hipFree(dev_c);

	return cudaStatus;
}
