#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "device_atomic_functions.h"

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <Windows.h>

#define TYPE unsigned long long
#define INT_64 unsigned long long

struct sJ {
	double s1, s4, s5, s6;
};

typedef struct progressData {
	volatile INT_64 *currentProgress;
	TYPE maxProgress;
	int quit = 0;
} PROGRESSDATA, *PPROGRESSDATA;

hipError_t addWithCuda(sJ *c, unsigned int size, TYPE digit);

//warpsize is 32 so optimal value is probably always a multiple of 32
const int threadCountPerBlock = 64;
//this is more difficult to optimize but seems to not like odd numbers
const int blockCount = 560;

__device__ const TYPE baseSystem = 16;
__device__ const int baseExpOf2 = 4;

__device__ const int typeSize = sizeof(TYPE) * 8 - 1;
__device__ const TYPE multiplyModCond = 0x2000000000000000;//2^61
__device__ const int int64Size = sizeof(INT_64) * 8 - 1;
__device__ const INT_64 int64ModCond = 0x40000000;
__device__ const INT_64 int64MaxBit = 0x8000000000000000;

__device__ int printOnce = 0;

//not actually quick
__device__ void quickMod(INT_64 input, const INT_64 mod, INT_64 *output) {

	/*INT_64 copy = input;
	INT_64 test = input % mod;*/
	INT_64 temp = mod;
	while (temp < input && !(temp&int64MaxBit)) temp <<= 1;
	if (temp > input) temp >>= 1;
	while (input >= mod && temp >= mod) {
		if(input >= temp) input -= temp;
		temp >>= 1;
	}
	/*if (input != test && !atomicAdd(&printOnce,1))
	{
		printf("input %llu mod %llu error\n", copy, mod);
		printOnce = 1;
	}*/
	*output = input;
}

//binary search to find highest 1 bit in multiplier
__device__ void findMultiplierHighestBit(const TYPE multiplier, TYPE *output) {
	
	//if no bits are 1 then highest bit doesn't exist
	if (!multiplier) {
		*output = 0;
		return;
	}

	int highestBitLocMax = typeSize;
	int highestBitLocMin = 0;

	int middle = (highestBitLocMax + highestBitLocMin) >> 1;

	TYPE highestBit = 1L;
	highestBit <<= middle;

	int less = highestBit <= multiplier;

	while (!((highestBit << 1) > multiplier && less)) {
		if (less) highestBitLocMin = middle + 1;
		else highestBitLocMax = middle - 1;
		middle = (highestBitLocMax + highestBitLocMin) >> 1;
		//this might not look necessary but it is
		highestBit = 1L;
		highestBit <<= middle;
		less = highestBit <= multiplier;
	}

	/*unsigned long long highestBit2 = 0x8000000000000000;

	while (highestBit2 > multiplier) highestBit2 >>= 1;

	if (highestBit != highestBit2 && !printOnce) {
		printf("multiplier %d error; highestBit %d; highestBit2 %d\n", multiplier, highestBit, highestBit2);
		printOnce = 1;
	}*/

	*output = highestBit;
}

//hacker's delight method to find highest bit in a long long (it just works)
//http://graphics.stanford.edu/~seander/bithacks.html
//just barely faster than built-in CUDA __clzll
__device__ void findMultiplierHighestBitHackersDelight(TYPE multiplier, TYPE *output) {
	
	multiplier |= multiplier >> 1;
	multiplier |= multiplier >> 2;
	multiplier |= multiplier >> 4;
	multiplier |= multiplier >> 8;
	multiplier |= multiplier >> 16;
	multiplier |= multiplier >> 32;

	*output = multiplier ^ (multiplier >> 1);

}

__device__ void modMultiplyLeftToRight(const TYPE multiplicand, const TYPE multiplier, TYPE mod, TYPE *output) {
	TYPE result = multiplicand;

	//TYPE highestBitMask = 0;

	//findMultiplierHighestBit(multiplier, &highestBitMask);

	/*unsigned long zeroCount = 0;

	zeroCount = __clzll(multiplier);*/

	TYPE highestBitMask = 0;

	//highestBitMask <<= (63 - zeroCount);
	
	findMultiplierHighestBitHackersDelight(multiplier, &highestBitMask);

	while (highestBitMask > 1) {
		//only perform modulus operation during loop if result is >= (TYPE maximum + 1)/8 (in order to prevent overflowing)
		if (result >= multiplyModCond) result %= mod;
		result <<= 1;
		highestBitMask >>= 1;
		if (multiplier&highestBitMask)	result += multiplicand;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//perform right-to-left binary exponention taking modulus of both base and result at each step
//64 bit integers are required to accurately find the modular exponents of numbers when mod is >= ~10e6
//however, with CUDA 64 bit integers are implemented at compile time as two 32 bit integers
//this produces about a 10x slowdown over computations using 32 bit integers
__device__ void modExp(unsigned long long base, long exp, long mod, long *output) {
	const unsigned long mask = 1;
	unsigned long long result = 1;

	//only perform modulus operation during loop if result or base is >= 2^32 (in order to prevent either from overflowing)
	//this saves 30% computation time over performing modulus in every loop iteration
	const unsigned long long modCond = 0x100000000;//2^32

	while (exp > 0) {
		if (exp&mask) {
			result *= base;
			if (result >= modCond) result %= mod;
		}
		base *= base;
		if (base >= modCond) base %= mod;
		exp >>= 1;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	result %= mod;
	*output = result;
}

//using left-to-right binary exponentiation
//the position of the highest bit in exponent is passed into the function as a parameter (it is more efficient to find it outside)
//this version allows base to be constant, thus reducing total number of moduli which must be calculated
//geometric mean of multiplication inputs is also substantially lower, allowing faster average multiplications
__device__ void modExpLeftToRight(const TYPE exp, TYPE mod, TYPE highestBitMask, TYPE *output) {
	INT_64 result = baseSystem;

	//only perform modulus operation during loop if result is >= sqrt((BIG_TYPE maximum + 1)/8) (in order to prevent overflowing)
	INT_64 modCond = int64ModCond;

	while (highestBitMask > 1) {
		//this is not necessary as modMultiplyLeftToRight ensures result never overflows a 64 bit buffer
		//however performing this modulus saves time (more is less)
		//likely saves performing some moduli in modMultiplyLeftToRight or reduces overall size of arguments
		if (result >= mod) result %= mod;//quickMod(result, mod, &result);

		modMultiplyLeftToRight(result, result, mod, &result);//result *= result;
		highestBitMask >>= 1;
		if (exp&highestBitMask)	result <<= baseExpOf2;//modMultiplyLeftToRight(result, base, mod, &result);//result *= base;
	}

	//modulus must be taken after loop as it hasn't necessarily been taken during last loop iteration
	//result %= mod;//quickMod(result, mod, &result);
	*output = result;
}

//find ( 16^n % mod ) / mod and add to partialSum
__device__ void fractionalPartOfSum(TYPE exp, TYPE mod, double *partialSum, TYPE highestBitMask) {
	TYPE expModResult = 0;
	modExpLeftToRight(exp, mod, highestBitMask, &expModResult);
	*partialSum += ((double)expModResult) / ((double)mod);
}

//stride over all parts of summation in bbp formula where k <= n
//to compute partial sJ sums
__device__ void bbp(TYPE n, TYPE start, TYPE stride, sJ *output, volatile INT_64 *progress) {

	double s1 = 0.0, s4 = 0.0, s5 = 0.0, s6 = 0.0;
	double trash = 0.0;
	TYPE highestExpBit = 1;
	while (highestExpBit <= n)	highestExpBit <<= 1;
	for (TYPE k = start; k <= n; k += stride) {
		while (highestExpBit > (n - k))  highestExpBit >>= 1;
		TYPE mod = 8 * k + 1;
		fractionalPartOfSum(n - k, mod, &s1, highestExpBit);
		mod += 3;
		fractionalPartOfSum(n - k, mod, &s4, highestExpBit);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s5, highestExpBit);
		mod += 1;
		fractionalPartOfSum(n - k, mod, &s6, highestExpBit);
		//remove any integer part of s1-s6
		s1 = modf(s1, &trash);
		s4 = modf(s4, &trash);
		s5 = modf(s5, &trash);
		s6 = modf(s6, &trash);
		if (start == 0) {
			//only 1 thread ever updates the progress
			*progress = k;
		}
	}
	output[start].s1 = s1;
	output[start].s4 = s4;
	output[start].s5 = s5;
	output[start].s6 = s6;
}

//determine from thread and block position where to begin stride
//and how wide stride is
__global__ void bbpKernel(sJ *c, volatile INT_64 *progress, TYPE digit)
{
	TYPE stride = blockDim.x * gridDim.x;
	TYPE i = threadIdx.x + blockDim.x * blockIdx.x;
	bbp(digit, i, stride, c, progress);
}

//stride over current leaves of reduce tree
__global__ void reduceSJKernel(sJ *c, int offset, int stop) {
	int stride = blockDim.x * gridDim.x;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	while (i < stop) {
		int augend = i + offset;
		c[i].s1 += c[augend].s1;
		c[i].s4 += c[augend].s4;
		c[i].s5 += c[augend].s5;
		c[i].s6 += c[augend].s6;
		i += stride;
	}
}

//standard tree-based parallel reduce
hipError_t reduceSJ(sJ *c, unsigned int size) {
	hipError_t cudaStatus;
	while (size > 1) {
		int nextSize = (size + 1) >> 1;

		//size is odd
		if (size&1) reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize - 1);
		//size is even
		else reduceSJKernel<< <32, 32 >> >(c, nextSize, nextSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "reduceSJKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching reduceSJKernel!\n", cudaStatus);
			return cudaStatus;
		}

		size = nextSize;
	}
	return cudaStatus;
}

//compute four steps of sJ sums for i > n and add to sJ sums found previously
//combine sJs according to bbp formula
//multiply by 16^5 to extract five digits of pi starting at n
long finalizeDigit(sJ input, TYPE n) {
	double reducer = 1.0;
	double s1 = input.s1;
	double s4 = input.s4;
	double s5 = input.s5;
	double s6 = input.s6;
	double trash = 0.0;
	if (n < 16000) {
		for (int i = 0; i < 4; i++) {
			n++;
			reducer /= (double)baseSystem;
			double eightN = 8.0 * n;
			s1 += reducer / (eightN + 1.0);
			s4 += reducer / (eightN + 4.0);
			s5 += reducer / (eightN + 5.0);
			s6 += reducer / (eightN + 6.0);
		}
	}
	//remove any integer part of s1-s6
	s1 = modf(s1, &trash);
	s4 = modf(s4, &trash);
	s5 = modf(s5, &trash);
	s6 = modf(s6, &trash);
	double hexDigit = 4.0*s1 - 2.0*s4 - s5 - s6;
	hexDigit = modf(hexDigit, &trash);
	if (hexDigit < 0) hexDigit++;
	hexDigit *= baseSystem*baseSystem*baseSystem*baseSystem*baseSystem;
	printf("hexDigit = %.8f\n", hexDigit);
	return (long)hexDigit;
}

int main()
{
	try {
		const int arraySize = threadCountPerBlock * blockCount;
		const TYPE digitPosition = 9999999999;
		sJ* c = new sJ[arraySize];

		clock_t start = clock();

		hipError_t cudaStatus = addWithCuda(c, arraySize, digitPosition);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}

		long hexDigit = finalizeDigit(c[0], digitPosition);

		clock_t end = clock();

		printf("pi at hexadecimal digit %llu is %X\n",
			digitPosition + 1, hexDigit);

		printf("Computed in %.8f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
			return 1;
		}

		return 0;
	}
	catch(...) {
		printf("oops xD\n");
		return 1;
	}
}

//this function is meant to be run by an independent thread to output progress to the console
DWORD WINAPI progressCheck(LPVOID data) {
	PPROGRESSDATA progP = (PPROGRESSDATA)data;

	double lastProgress = 0;

	while(!(*progP).quit) {
		double progress = (double)(*((*progP).currentProgress)) / (double)(*progP).maxProgress;

		if (progress - lastProgress > 1E-4) {
			double timeEst = (1.0 - progress)*0.1 / (progress - lastProgress);
			lastProgress = progress;
			printf("Current progress is %3.3f%%. Estimated total runtime remaining is %8.3f seconds.\n", 100.0*progress, timeEst);
		}

		Sleep(100);
	}
	return 0;
}

// Helper function for using CUDA
hipError_t addWithCuda(sJ *c, unsigned int size, TYPE digit)
{
	sJ *dev_c = 0;

	//these variables are linked between host and device memory allowing each to communicate about progress
	volatile INT_64 *currProgHost, *currProgDevice;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//allow device to map host memory for progress ticker
	hipSetDeviceFlags(hipDeviceMapHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDeviceFlags failed!");
		goto Error;
	}

	// Allocate GPU buffer for output vector    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(sJ));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate Host memory for progress ticker
	cudaStatus = hipHostAlloc((void**)&currProgHost, sizeof(INT_64), hipHostMallocMapped);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaHostAalloc failed!");
		goto Error;
	}

	//create link between between host and device memory for progress ticker
	cudaStatus = hipHostGetDevicePointer((INT_64 **)&currProgDevice, (INT_64 *)currProgHost, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipHostGetDevicePointer failed!");
		goto Error;
	}

	*currProgHost = 0;

	PROGRESSDATA threadData = { currProgHost, digit, 0 };

	HANDLE thread = CreateThread(NULL, 0, *progressCheck, (LPVOID) &threadData, 0, NULL);

	if (thread == NULL) {
		fprintf(stderr, "progressCheck thread creation failed\n");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	bbpKernel << <blockCount, threadCountPerBlock >> >(dev_c, currProgDevice, digit);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "bbpKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching bbpKernel!\n", cudaStatus);
		goto Error;
	}

	//tell the progress thread to quit
	threadData.quit = 1;

	WaitForSingleObject(thread, INFINITE);
	CloseHandle(thread);

	cudaStatus = reduceSJ(dev_c, size);

	if (cudaStatus != hipSuccess) {
		goto Error;
	}

	// Copy result vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(sJ), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);

	return cudaStatus;
}
